#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include "ppl/kernel/llm/cuda/xformer/fmha.h"
#include "utils/kernel_forward.h"

#include "autogen/cutlassF.h"

#include <hip/hip_fp16.h>
#include <cmath>

namespace ppl { namespace kernel { namespace llm { namespace cuda { namespace xformer {

struct FmhaKernelHelper {
    template<typename KT, typename FT>
    void operator()(KT _k, FT kernel_fn) {
        using Kernel = decltype(_k);
        using scalar_t = typename Kernel::scalar_t;
        (void)_k;

        if (kernel_launched) {
            kernel_miss_reason = "kernel launched";
            return;
        }

        if (!Kernel::kSupportsBias && (optional_attn_mask != nullptr)) {
            kernel_miss_reason = "xformer kernel does not support bias";
            return;
        }

        if (Kernel::kSingleValueIteration && Kernel::kKeysPerBlock < head_dim) {
            kernel_miss_reason = "xformer kernel does not support head_dim";
            return;
        }

        // Uses too much shmem
        size_t smem_bytes = sizeof(typename Kernel::SharedStorage);
        if (smem_bytes > max_shmem) {
            kernel_miss_reason = "xformer kernel use too much shm";
            return;
        }

        typename Kernel::Params p;
        p.query_ptr = (scalar_t*)query;
        p.key_ptr = (scalar_t*)key;
        p.value_ptr = (scalar_t*)value;

        p.logsumexp_ptr = nullptr;
        p.output_accum_ptr = nullptr;

        p.output_ptr = (typename Kernel::output_t*)output;

        if (optional_seqstart_q != nullptr) {
            p.seqstart_q_ptr = (int64_t*)optional_seqstart_q;
            p.seqstart_k_ptr = (int64_t*)optional_seqstart_k;
        }

        p.num_heads = num_heads;
        p.num_kv_repeats = num_heads / num_kv_heads;
        p.head_dim = head_dim;
        p.head_dim_value = head_dim;
        p.num_queries = max_seqlen;
        p.num_keys = optional_seqstart_q == nullptr ? max_kvlen : 0;
        p.num_batches = batch;
        p.custom_mask_type = custom_mask_type;
        p.seqlen_k_ptr = nullptr;

        if (attn_scale != 0) {
            p.scale = attn_scale;
        } else {
            p.scale = 1.0f / std::sqrt(float(p.head_dim));
        }

        p.q_strideB = query_stride_b;
        p.k_strideB = key_stride_b;
        p.v_strideB = value_stride_b;

        p.q_strideM = query_stride_s;
        p.k_strideM = key_stride_s;
        p.v_strideM = value_stride_s;

        p.q_strideH = query_stride_h;
        p.k_strideH = key_stride_h;
        p.v_strideH = value_stride_h;

        p.o_strideM = output_stride_s;

        if (optional_attn_mask != nullptr) {
            p.attn_bias_ptr = (scalar_t*)optional_attn_mask;
            p.bias_strideB = mask_stride_b;
            p.bias_strideH = mask_stride_h;
            p.bias_strideM = mask_stride_s;
        }

        p.use_dropout = false;

        if (smem_bytes > 0xc000) {
            auto err = hipFuncSetAttribute(reinterpret_cast<const void*>(
                kernel_fn), hipFuncAttributeMaxDynamicSharedMemorySize, smem_bytes);
            if (err == hipErrorInvalidValue) {
                kernel_miss_reason = "this GPU does not have enough shared-memory kernel requires";
                return;
            }
        }

        if(!Kernel::check_supported(p)) {
            kernel_miss_reason = "xformer get unsupported param";
            return;
        }

        kernel_fn<<<p.getBlocksGrid(), p.getThreadsGrid(), smem_bytes, stream>>>(p);
        kernel_launched = true;
    }

    hipStream_t stream;
    const void* query;
    const void* key;
    const void* value;
    const void* optional_attn_mask;
    const void* optional_seqstart_q; // (B + 1)
    const void* optional_seqstart_k; // (B + 1)
    int64_t batch;
    int64_t query_stride_b; // 0 if dynamic batch
    int64_t query_stride_s;
    int64_t query_stride_h;
    int64_t key_stride_b; // 0 if dynamic batch
    int64_t key_stride_s;
    int64_t key_stride_h;
    int64_t value_stride_b; // 0 if dynamic batch
    int64_t value_stride_s;
    int64_t value_stride_h;
    int64_t mask_stride_b; // 0 if dynamic batch
    int64_t mask_stride_s;
    int64_t mask_stride_h;
    int64_t output_stride_s;
    int64_t max_seqlen;
    int64_t max_kvlen; // unused if dynamic batch
    int64_t num_heads;
    int64_t num_kv_heads;
    int64_t head_dim;
    int64_t custom_mask_type;
    float attn_scale;
    void* output;

    size_t max_shmem;
    int compute_capability;

    bool &kernel_launched;
    const char *&kernel_miss_reason;
};

ppl::common::RetCode fmha(
    const hipStream_t stream,
    const hipDeviceProp_t& device_prop,
    const ppl::common::datatype_t datatype,
    const void* query,
    const void* key,
    const void* value,
    const void* optional_attn_mask,
    const void* optional_seqstart_q, // (B + 1)
    const void* optional_seqstart_k, // (B + 1)
    const int64_t batch,
    const int64_t query_stride_b, // 0 if dynamic batch
    const int64_t query_stride_s,
    const int64_t query_stride_h,
    const int64_t key_stride_b, // 0 if dynamic batch
    const int64_t key_stride_s,
    const int64_t key_stride_h,
    const int64_t value_stride_b, // 0 if dynamic batch
    const int64_t value_stride_s,
    const int64_t value_stride_h,
    const int64_t mask_stride_b, // 0 if dynamic batch
    const int64_t mask_stride_s,
    const int64_t mask_stride_h,
    const int64_t output_stride_s,
    const int64_t max_seqlen,
    const int64_t max_kvlen, // unused if dynamic batch
    const int64_t num_heads,
    const int64_t num_kv_heads,
    const int64_t head_dim,
    const int64_t custom_mask_type,
    const float attn_scale,
    void* output)
{
    if (datatype != ppl::common::DATATYPE_FLOAT16) {
        LOG(ERROR) << "only support fp16";
        return ppl::common::RC_UNSUPPORTED;
    }

    bool kernel_launched = false;
    const char *kernel_miss_reason = nullptr;

    const int compute_capability = device_prop.major * 10 + device_prop.minor;
    const auto max_shmem = device_prop.sharedMemPerBlockOptin;

    FmhaKernelHelper hlp = {
        stream,
        query,
        key,
        value,
        optional_attn_mask,
        optional_seqstart_q, // (B + 1)
        optional_seqstart_k, // (B + 1)
        batch,
        query_stride_b, // 0 if dynamic batch
        query_stride_s,
        query_stride_h,
        key_stride_b, // 0 if dynamic batch
        key_stride_s,
        key_stride_h,
        value_stride_b, // 0 if dynamic batch
        value_stride_s,
        value_stride_h,
        mask_stride_b, // 0 if dynamic batch
        mask_stride_s,
        mask_stride_h,
        output_stride_s,
        max_seqlen,
        max_kvlen, // unused if dynamic batch
        num_heads,
        num_kv_heads,
        head_dim,
        custom_mask_type,
        attn_scale,
        output,
        // other param
        max_shmem,
        compute_capability,
        // ret param
        kernel_launched,
        kernel_miss_reason,
    };

    dispatch_cutlassF<::cutlass::half_t>(hlp, compute_capability);

    if (!kernel_launched) {
        LOG(ERROR) << "xformer kernel not launched, reason: " << kernel_miss_reason;
        return ppl::common::RC_UNSUPPORTED;
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        LOG(ERROR) << "CUDA Error: " << hipGetErrorString(err);
        return ppl::common::RC_DEVICE_RUNTIME_ERROR;
    }

    return ppl::common::RC_SUCCESS;
}

}}}}}
