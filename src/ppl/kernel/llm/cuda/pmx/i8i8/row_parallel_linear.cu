#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include "ppl/kernel/llm/cuda/pmx/row_parallel_linear.h"
#include "ppl/common/log.h"

#include "ppl/kernel/llm/cuda/pmx/i8i8/quantize.h"

namespace ppl { namespace kernel { namespace llm { namespace cuda { namespace pmx { namespace i8i8 {

// input should be m*k, CUBLASLT_ORDER_COL32
// weight should be n*k, CUBLASLT_ORDER_COL32_2R_4R4 or CUBLASLT_ORDER_COL4_4R2_8C
// output is m*n, CUBLASLT_ORDER_COL32
ppl::common::RetCode row_parallel_linear(
    const hipStream_t stream,
    const hipblasLtHandle_t& cublaslt_handle,
    const hipblasLtMatmulAlgo_t* algo,
    const ppl::common::TensorShape* input_shape,
    const void* input,
    const ppl::common::TensorShape* weight_shape,
    const void* weight,
    const ppl::common::TensorShape* bias_shape,
    const void* bias,
    const void* scale_M,
    const void* scale_N,
    const float down_scale_M,
    const float down_scale_N,
    const int64_t in_features,
    const int64_t out_features,
    const matrix_layout_t weight_layout,
    const ppl::common::NcclParam* nccl_param,
    const bool input_is_parallel,
    void* split_buffer,
    void* quant_buffer,
    const int64_t cublas_workspace_size,
    void* cublas_workspace,
    ppl::kernel::llm::cuda::cublas::AlgoCache* cublas_algo_cache,
    const ppl::common::TensorShape* output_shape,
    void* output)
{
    if (!input_is_parallel) {
        LOG(ERROR) << "currnetly only support parallel input";
        return ppl::common::RC_UNSUPPORTED;
    }

    if (bias && bias_shape->GetDataType() != ppl::common::DATATYPE_FLOAT16) {
        LOG(ERROR) << "only support fp16 bias";
        return ppl::common::RC_UNSUPPORTED;
    }

    if (weight_layout != MATRIX_LAYOUT_ROW_MAJOR &&
        weight_layout != MATRIX_LAYOUT_COL4_4R2_8C &&
        weight_layout != MATRIX_LAYOUT_COL32_2R_4R4)
    {
        LOG(ERROR) << "unsupported weight layout:" << (int32_t)weight_layout;
        return ppl::common::RC_UNSUPPORTED;
    }

    const bool use_col32_gemm = weight_layout != MATRIX_LAYOUT_ROW_MAJOR;
    const bool use_4r4_weight = weight_layout == MATRIX_LAYOUT_COL32_2R_4R4;

    // input (M, K/w)
    // weight (N, K/w)
    // output (M, N)

    const int64_t M = input_shape->CalcElementsToDimensionExcludingPadding(input_shape->GetDimCount() - 1);
    const int64_t N = out_features;
    const int64_t Kw = in_features / nccl_param->size;

    const void* reduce_bias = nccl_param->rank == 0 ? bias : nullptr;

    ppl::common::RetCode status = ppl::common::RC_SUCCESS;

    // LOG(ERROR) << "M" << M << ", N" << N << ", K" << Kw;

    if (!use_col32_gemm) {
        status = ppl::kernel::llm::cuda::cublas::gemm_i8i8i32(
            stream,
            cublaslt_handle,
            algo,
            false,
            Kw,
            input_shape->GetDataType(),
            input,
            true,
            Kw,
            weight_shape->GetDataType(),
            weight,
            nullptr,
            M,
            N,
            Kw,
            1,
            0,
            cublas_workspace_size,
            cublas_workspace,
            cublas_algo_cache,
            N,
            ppl::common::DATATYPE_INT32,
            quant_buffer);
    } else {
        status = ppl::kernel::llm::cuda::cublas::gemm_i8i8i32_col32(
            stream,
            cublaslt_handle,
            input,
            weight,
            M,
            N,
            Kw,
            use_4r4_weight,
            quant_buffer);
    }

    if (ppl::common::RC_SUCCESS != status)
        return status;

    status = ppl::kernel::llm::cuda::pmx::i8i8::minmax_dequantize_fp16(
        stream,
        quant_buffer,
        reduce_bias,
        scale_M,
        scale_N,
        M,
        N,
        down_scale_M,
        down_scale_N,
        (
            use_col32_gemm ?
            MATRIX_LAYOUT_COL32 :
            MATRIX_LAYOUT_ROW_MAJOR
        ),
        output
    );

    if (ppl::common::RC_SUCCESS != status)
        return status;

    if (nccl_param->size > 1) {
        return ppl::common::NcclAllReduceSum<half>(
            (half*)output,
            (half*)output,
            M * N,
            nccl_param,
            stream);
    }

    return ppl::common::RC_SUCCESS;
}

}}}}}}
