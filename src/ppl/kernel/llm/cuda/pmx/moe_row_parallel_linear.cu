#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include "ppl/kernel/llm/cuda/pmx/moe_row_parallel_linear.h"
#include "ppl/common/log.h"

#include <hip/hip_fp16.h>

namespace ppl { namespace kernel { namespace llm { namespace cuda { namespace pmx {

ppl::common::RetCode moe_row_parallel_linear(
    const hipStream_t stream,
    const hipblasLtHandle_t& cublaslt_handle,
    const hipblasLtMatmulAlgo_t* algo,
    const ppl::common::TensorShape* input_shape,
    const void* input,
    const ppl::common::TensorShape* weight_shape,
    const void* weight,
    const ppl::common::TensorShape* bias_shape,
    const void* bias,
    const ppl::common::TensorShape* offset_shape,
    const void* expert_offset,
    const int64_t in_features,
    const int64_t out_features,
    const ppl::common::NcclParam* nccl_param,
    const bool input_is_parallel,
    void* split_buffer,
    const int64_t cublas_workspace_size,
    void* cublas_workspace,
    const ppl::common::TensorShape* output_shape,
    void* output)
{
    if (!input_is_parallel) {
        LOG(ERROR) << "currnetly only support parallel input";
        return ppl::common::RC_UNSUPPORTED;
    }

    // input [seqlen * num_experts_per_token, hidden_dim/w]
    // weight [num_experts_per_token, hidden_dim_out, hidden_dim/w]
    // output [seqlen * num_experts_per_token, hidden_dim_out]

    const int64_t M = input_shape->CalcElementsToDimensionExcludingPadding(input_shape->GetDimCount() - 1);
    const int64_t N = out_features;
    const int64_t Kw = in_features / nccl_param->size;
    const int64_t* offset64_ptr = (const int64_t*)expert_offset;
    const int64_t num_experts = weight_shape->GetDim(0);
    const void *bias_ = nullptr;
    ppl::common::RetCode status;
    for (int i = 0; i < num_experts; ++i) {
        const int64_t start = offset64_ptr[i];
        const int64_t end = offset64_ptr[i + 1];
        if (end - start <= 0) {
            continue;
        }
        if (bias != nullptr) {
            bias_ = (char*)bias + start * N * ppl::common::GetSizeOfDataType(bias_shape->GetDataType());
        }
        const void *input_ = (char*)input + start * Kw * ppl::common::GetSizeOfDataType(input_shape->GetDataType());
        const void *weight_ = (char*)weight + start * N * Kw * ppl::common::GetSizeOfDataType(weight_shape->GetDataType());
        void *gemm_output_ = output + start * N * ppl::common::GetSizeOfDataType(output_shape->GetDataType());
        status = ppl::kernel::llm::cuda::cublas::gemm(
            stream,
            cublaslt_handle,
            algo,
            false,
            Kw,
            input_shape->GetDataType(),
            input_,
            true,
            Kw,
            weight_shape->GetDataType(),
            weight_,
            bias_,
            end - start,
            N,
            Kw,
            1.0f,
            0.0f,
            cublas_workspace_size,
            cublas_workspace,
            N,
            output_shape->GetDataType(),
            gemm_output_);
    
    }


    if (ppl::common::RC_SUCCESS != status)
        return status;

    if (nccl_param->size > 1) {
        return ppl::common::NcclAllReduceSum<half>(
            (half*)output,
            (half*)output,
            M * N,
            nccl_param,
            stream);
    }

    return ppl::common::RC_SUCCESS;
}

}}}}}
