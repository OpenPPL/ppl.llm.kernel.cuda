#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include "ppl/kernel/llm/cuda/pmx/linear.h"
#include "ppl/common/log.h"

#include "cudakernel/memory/transpose.h"

#include <hip/hip_fp16.h>

namespace ppl { namespace kernel { namespace llm { namespace cuda { namespace pmx {

ppl::common::RetCode linear(
    const hipStream_t stream,
    const hipblasLtHandle_t& cublaslt_handle,
    const hipblasLtMatmulAlgo_t* algo,
    const ppl::common::TensorShape* input_shape,
    const void* input,
    const ppl::common::TensorShape* weight_shape,
    const void* weight,
    const ppl::common::TensorShape* bias_shape,
    const void* bias,
    const int64_t in_features,
    const int64_t out_features,
    const int64_t cublas_workspace_size,
    void* cublas_workspace,
    const ppl::common::TensorShape* output_shape,
    void* output) 
{
    // input (M, K)
    // weight (N, K)
    // output (M, N)

    const int64_t M = input_shape->CalcElementsToDimensionExcludingPadding(input_shape->GetDimCount() - 1);
    const int64_t N = out_features;
    const int64_t K = in_features;

    void *gemm_output = output;

    return ppl::kernel::llm::cuda::cublas::gemm(
        stream,
        cublaslt_handle,
        algo,
        false,
        K,
        input_shape->GetDataType(),
        input,
        true,
        K,
        weight_shape->GetDataType(),
        weight,
        bias,
        M,
        N,
        K,
        1.0f,
        0.0f,
        cublas_workspace_size,
        cublas_workspace,
        N,
        output_shape->GetDataType(),
        gemm_output);
}

}}}}}