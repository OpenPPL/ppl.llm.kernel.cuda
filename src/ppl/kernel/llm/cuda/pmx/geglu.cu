#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include "ppl/kernel/llm/cuda/pmx/geglu.h"
#include "ppl/common/log.h"

#include <hip/hip_fp16.h>

namespace ppl { namespace kernel { namespace llm { namespace cuda { namespace pmx {

template<bool APPROXIMATE>
__global__ void geglu_kernel_fp16(
    const half *input,
    const int64_t batch,
    const int64_t num_elem,
    half *output
) {
    const int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= batch * num_elem)
        return;

    const int64_t b = index / num_elem;
    const int64_t i = index % num_elem;

    auto val = __half2float(input[(b * 2) * num_elem + i]);
    auto gate_val = input[(b * 2 + 1) * num_elem + i];

    float out_val = 0.f;
    if (APPROXIMATE) {
        out_val = val * 0.5f * (1.f + tanh(0.7978845608028654f * val * (1.0f + 0.044715f * val * val)));
    } else {
        out_val = val * 0.5f * (1.f + erff(val * 0.707106781f));
    }
    output[index] = __float2half(out_val) * gate_val;
}

template<bool APPROXIMATE>
__global__ void geglu_kernel_packed_fp16(
    const half2 *input,
    const int64_t batch,
    const int64_t num_elem,
    half2 *output)
{
    const int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= batch * num_elem)
        return;

    const int64_t b = index / num_elem;
    const int64_t i = index % num_elem;

    auto h_val = input[(b * 2) * num_elem + i];
    auto f_val = __half22float2(input[(b * 2) * num_elem + i]);
    auto gate_val = input[(b * 2 + 1) * num_elem + i];

    half2 t_val;
    if (APPROXIMATE) {
        t_val.x = __float2half(tanh(0.7978845608028654f * f_val.x * (1.0f + 0.044715f * f_val.x * f_val.x)));
        t_val.y = __float2half(tanh(0.7978845608028654f * f_val.y * (1.0f + 0.044715f * f_val.y * f_val.y)));
    } else {
        t_val.x = __float2half(erff(f_val.x * 0.707106781f));
        t_val.y = __float2half(erff(f_val.y * 0.707106781f));
    }
    
    half2 one_constant = {__float2half(1.f),  __float2half(1.f)};
    half2 half_constant = {__float2half(0.5f),  __float2half(0.5f)};
    t_val = __hmul2(half_constant, __hmul2(h_val, __hadd2(one_constant, t_val)));

    output[index] = {
        t_val.x * gate_val.x,
        t_val.y * gate_val.y,
    };
}

ppl::common::RetCode geglu(
    hipStream_t stream,
    const void* input,
    const bool approximate,
    const ppl::common::TensorShape* output_shape,
    void* output)
{
    if (output_shape->GetDataType() != ppl::common::DATATYPE_FLOAT16) {
        LOG(ERROR) << "geglu only support fp16, but got ["<< output_shape->GetDataType() << "]";
        return ppl::common::RC_UNSUPPORTED;
    }

    const int64_t TPB = 256;    // thread_per_block
    const int64_t num_elem = output_shape->GetDim(output_shape->GetDimCount() - 1);
    const int64_t batch = output_shape->CalcElementsIncludingPadding() / num_elem;

    if (num_elem & 1) {
        const int64_t BPG = ((batch * num_elem) + TPB - 1) / TPB; // block_per_grid
        if (approximate) {
            geglu_kernel_fp16<true><<<BPG, TPB, 0, stream>>>(
                (const half*)input, batch, num_elem, (half*)output);
        } else {
            geglu_kernel_fp16<false><<<BPG, TPB, 0, stream>>>(
                (const half*)input, batch, num_elem, (half*)output);
        }

    } else {
        const int64_t BPG = (((batch * num_elem) >> 1) + TPB - 1) / TPB;
        if (approximate) {
            geglu_kernel_packed_fp16<true><<<BPG, TPB, 0, stream>>>(
                (const half2*)input, batch, num_elem, (half2*)output);
        } else {
            geglu_kernel_packed_fp16<false><<<BPG, TPB, 0, stream>>>(
                (const half2*)input, batch, num_elem, (half2*)output);
        }
    }

    return ppl::common::RC_SUCCESS;
}


}}}}}
