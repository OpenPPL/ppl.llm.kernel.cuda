#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include "ppl/kernel/llm/cuda/pmx/sample.h"
#include "ppl/common/log.h"

#include "cudakernel/common/common.cuh"

#include <hip/hip_runtime.h>
#include <float.h>
#include <hipcub/hipcub.hpp>

namespace ppl { namespace kernel { namespace llm { namespace cuda { namespace pmx {

using fp32_t = float;

struct SortingPair {
    fp32_t value;
    int32_t index;
    __device__ SortingPair(fp32_t value, int32_t index): value(value), index(index) {}
};

template<typename Dtype, int32_t TPB>
__device__ __host__ inline
int32_t pad_vocab(int32_t vocab_size)
{
    // for vector load/store
    constexpr int32_t VPT = 16 / sizeof(Dtype);
    return vocab_size + (TPB * VPT) - vocab_size % (TPB * VPT);
}

template <int32_t TPB, int32_t VPT>
struct CachedVocabStorage {
    fp32_t* local_storage;
    const fp32_t *data_ptr;
    int32_t index;

    __device__ inline CachedVocabStorage(
        const fp32_t *data_ptr,
        fp32_t *local_storage,
        const int64_t base_idx)
    {
        this->data_ptr      = data_ptr;
        this->local_storage = local_storage;
        this->index         = base_idx;
    }

    __device__ inline fp32_t Pop(int32_t local_selection)
    {
        if(local_selection % VPT == 0)
            copy<sizeof(float) * VPT>(data_ptr + index + local_selection, local_storage);
        return local_storage[local_selection % VPT];
    }
};

template<int32_t WPT>
__device__ inline
fp32_t sample_block_reduce_max(fp32_t reducing, fp32_t *shared_mem)
{
    // Helper function for reduce max.
    constexpr int32_t WARP_SIZE = 32;
    const int32_t lane_id = threadIdx.x % WARP_SIZE;
    const int32_t warp_id = threadIdx.x / WARP_SIZE;

    for (int32_t mask = WARP_SIZE / 2; mask >= 1; mask /= 2) {
        reducing = fmaxf(reducing, __shfl_xor_sync(uint32_t(-1), reducing, mask));
    }

    if (lane_id == 0) {
        shared_mem[warp_id] = reducing;
    }
    __syncthreads();

    if (lane_id < WPT) reducing = shared_mem[lane_id];
    else reducing = -FLT_MAX;

# pragma unroll
    for (int32_t mask = WPT / 2; mask >= 1; mask /= 2) {
        reducing = fmaxf(reducing, __shfl_xor_sync(uint32_t(-1), reducing, mask));
    }

    reducing = __shfl_sync(uint32_t(-1), reducing, 0);
    return reducing;
}

template<int32_t WPT>
__device__ inline
SortingPair sample_block_reduce_max_with_index(fp32_t reducing, int32_t index, void *shared_mem)
{
    // Helper function for reduce max.
    constexpr int32_t WARP_SIZE = 32;
    const int32_t lane_id = threadIdx.x % WARP_SIZE;
    const int32_t warp_id = threadIdx.x / WARP_SIZE;

    int32_t* shared_mem_i32 = reinterpret_cast<int32_t*>(shared_mem);
    fp32_t* shared_mem_fp32 = reinterpret_cast<fp32_t*>(shared_mem) + WPT;

    fp32_t reducing_value = reducing, receving_value;
    int32_t reducing_index = index, receving_index;

    for (int32_t mask = WARP_SIZE / 2; mask >= 1; mask /= 2) {
        receving_value = __shfl_xor_sync(uint32_t(-1), reducing_value, mask);
        receving_index = __shfl_xor_sync(uint32_t(-1), reducing_index, mask);

        if (receving_value > reducing_value){
            reducing_value = receving_value;
            reducing_index = receving_index;
        }
    }

    if (lane_id == 0) {
        shared_mem_fp32[warp_id] = reducing_value;
        shared_mem_i32[warp_id] = reducing_index;
    }
    __syncthreads();

    if (lane_id < WPT) {
        reducing_value = shared_mem_fp32[lane_id];
        reducing_index = shared_mem_i32[lane_id];
    }
    else {
        reducing_value = -FLT_MAX;
        reducing_index = -1;
    }

# pragma unroll
    for (int32_t mask = WPT / 2; mask >= 1; mask /= 2) {
        receving_value = __shfl_xor_sync(uint32_t(-1), reducing_value, mask);
        receving_index = __shfl_xor_sync(uint32_t(-1), reducing_index, mask);

        if (receving_value > reducing_value){
            reducing_value = receving_value;
            reducing_index = receving_index;
        }
    }

    reducing_value = __shfl_sync(uint32_t(-1), reducing_value, 0);
    reducing_index = __shfl_sync(uint32_t(-1), reducing_index, 0);
    return SortingPair(reducing_value, reducing_index);
}


template<int32_t WPT>
__device__ inline
fp32_t sample_block_reduce_sum(fp32_t reducing, fp32_t *shared_mem)
{
    // Helper function for reduce sum.
    constexpr int32_t WARP_SIZE = 32;
    const int32_t lane_id = threadIdx.x % WARP_SIZE;
    const int32_t warp_id = threadIdx.x / WARP_SIZE;

# pragma unroll
    for (int32_t mask = WARP_SIZE / 2; mask >= 1; mask /= 2) {
        reducing += __shfl_xor_sync(uint32_t(-1), reducing, mask);
    }

    if (lane_id == 0) shared_mem[warp_id] = reducing;
    __syncthreads();

    if (lane_id < WPT) reducing = shared_mem[lane_id];

# pragma unroll
    for (int32_t mask = WPT / 2; mask >= 1; mask /= 2) {
        reducing += __shfl_xor_sync(uint32_t(-1), reducing, mask);
    }
    reducing = __shfl_sync(uint32_t(-1), reducing, 0);
    return reducing;
}

struct SamplePrefixOp
{
    // Powered by TRT
    fp32_t running_total;

    __device__ SamplePrefixOp(fp32_t running_total) : running_total(running_total) {}

    __device__ fp32_t operator() (fp32_t block_aggregate) {
        fp32_t old_prefix = running_total;
        running_total += block_aggregate;
        return old_prefix;
    }
};

template<int32_t TPB, int32_t VPT>
__global__
void sample_topk_topp_default_kernel(
    const fp32_t __restrict__ *logits,      // [num_batches, batch_stride]
    const fp32_t *temperatures,             // [num_batches]
    const fp32_t *top_p,                    // [num_batches]
    const fp32_t *rnd,                      // [num_batches]
    const int32_t vocab_size,
    const int32_t batch_stride,
    const int32_t top_k_val,
    const fp32_t top_p_val,
    const fp32_t rnd_val,
    fp32_t *padded_logits,                  // [num_batches, padded_vocab_size]
    int32_t *output)                        // [num_batches, 1])
{
    constexpr int32_t WARP_SIZE = 32;
    constexpr int32_t WPT = TPB / WARP_SIZE; // warp per thread block.

    const int64_t batch_id            = blockIdx.x;
    const int64_t batch_offset        = batch_id * batch_stride;
    const int64_t padded_batch_offset = batch_id * pad_vocab<fp32_t, TPB>(vocab_size);
    const fp32_t  temperature         = temperatures ? max(abs(temperatures[batch_id]) + 1e-7, 0.01): 1.0f; // temperature 最低 0.01

    extern __shared__ fp32_t topk_shm[];
    fp32_t *topk_sums = topk_shm;
    int32_t *topk_idxs = reinterpret_cast<int32_t*>(&topk_shm[top_k_val]);
    __shared__ fp32_t reducing_memory[WPT * 2];

    fp32_t max_val = -FLT_MAX;
    int32_t max_idx = -1;
    float local_sum = 1.f;
    float local_top_p = top_p == nullptr ? top_p_val : top_p[batch_id];

    for (int32_t vocab_base = threadIdx.x * VPT; vocab_base < vocab_size; vocab_base += TPB * VPT) {
        fp32_t local_vals[VPT];

        #pragma unroll
        for (int32_t vec_idx = 0; vec_idx < VPT; vec_idx++) {
            int32_t vocab_idx = vocab_base + vec_idx;
            // TODO: optimize branch (vocab_idx < vocab_size)
            local_vals[vec_idx] = vocab_idx < vocab_size
                ? logits[batch_offset + vocab_idx]
                : -FLT_MAX;
            if (local_vals[vec_idx] > max_val) {
                max_val = local_vals[vec_idx];
                max_idx = vocab_idx;
            }
        }

        // TODO: optimize branch
        if (local_top_p != 0.f)
            copy<VPT * sizeof(fp32_t)>(local_vals, &padded_logits[padded_batch_offset + vocab_base]);
    }
    SortingPair max_pair = sample_block_reduce_max_with_index<WPT>(max_val, max_idx, reducing_memory);
    max_val = max_pair.value;
    max_idx = max_pair.index;

    if (local_top_p == 0.f) {
        if (threadIdx.x == 0) {
            output[batch_id] = max_idx;
        }
    } else {
        if (threadIdx.x == 0) {
            topk_sums[0] = 1.f;
            topk_idxs[0] = max_idx;
            padded_logits[padded_batch_offset + max_idx] = -FLT_MAX;
        }
        __syncthreads();

        for (int32_t i = 1; i < top_k_val; i++) {
            fp32_t local_max_val = -FLT_MAX;
            int32_t local_max_idx = -1;
            for (int32_t vocab_base = threadIdx.x * VPT; vocab_base < vocab_size; vocab_base += TPB * VPT) {
                fp32_t local_vals[VPT];
                copy<VPT * sizeof(fp32_t)>(&padded_logits[padded_batch_offset + vocab_base], local_vals);

                #pragma unroll
                for (int32_t vec_idx = 0; vec_idx < VPT; vec_idx++) {
                    if (local_vals[vec_idx] > local_max_val) {
                        local_max_val = local_vals[vec_idx];
                        local_max_idx = vocab_base + vec_idx;
                    }
                }
            }
            SortingPair p = sample_block_reduce_max_with_index<WPT>(local_max_val, local_max_idx, reducing_memory);
            local_max_val = p.value;
            local_max_idx = p.index;

            local_max_val = exp((local_max_val - max_val) / temperature);
            local_sum += local_max_val;

            if (threadIdx.x == 0) {
                topk_sums[i] = local_sum;
                topk_idxs[i] = local_max_idx;
                padded_logits[padded_batch_offset + local_max_idx] = -FLT_MAX;
            }
        }

        __syncthreads();

        local_sum = __fdividef(1.f, local_sum + 1e-6f);
        const fp32_t top_p_selection = local_top_p * (rnd == nullptr ? rnd_val : rnd[batch_id]);
        fp32_t prob_sum = (threadIdx.x < top_k_val) ? topk_sums[threadIdx.x] * local_sum : 1.f;
        int32_t count = __syncthreads_count((int32_t)(prob_sum >= top_p_selection));

        if (threadIdx.x == min(TPB - count, TPB - 1)) {
            output[batch_id] = topk_idxs[threadIdx.x];
        }
    }
}


template<int32_t TPB, int32_t LPT, int32_t KPT>
__global__
void sample_topk_topp_radix_select_kernel(
    const fp32_t __restrict__ *logits,      // [num_batches, batch_stride]
    const fp32_t *temperatures,             // [num_batches]
    const fp32_t *top_p,                    // [num_batches]
    const fp32_t *rnd,                      // [num_batches]
    const int32_t vocab_size,
    const int32_t batch_stride,
    const int32_t top_k_val,
    const fp32_t top_p_val,
    const fp32_t rnd_val,
    int32_t *output)                        // [num_batches, 1])
{
    constexpr int32_t WARP_SIZE = 32;
    constexpr int32_t WPT = TPB / WARP_SIZE; // warp per thread block.
    constexpr int32_t LOAD_LEN = LPT - KPT;

    const int64_t batch_id            = blockIdx.x;
    const int64_t batch_offset        = batch_id * batch_stride;
    const fp32_t  temperature         = temperatures ? max(abs(temperatures[batch_id]) + 1e-7, 0.01): 1.0f; // temperature 最低 0.01

    __shared__ fp32_t reducing_memory[WPT * 2];

    fp32_t local_vals[LPT];
    int32_t local_idxs[LPT];
    #pragma unroll
    for (int32_t i = 0; i < KPT; i++) {
        int32_t vocab_idx = threadIdx.x + i * TPB;
        if (vocab_idx < vocab_size) {
            local_vals[i] = logits[batch_offset + vocab_idx];
            local_idxs[i] = vocab_idx;
        } else {
            local_vals[i] = -FLT_MAX;
            local_idxs[i] = -1;
        }
    }

    typedef hipcub::BlockRadixSort<fp32_t, TPB, LPT, int32_t> BlockRadixSort;
    __shared__ typename BlockRadixSort::TempStorage sort_storage;

    for (int32_t vocab_base = threadIdx.x + KPT * TPB; vocab_base < vocab_size; vocab_base += TPB * LOAD_LEN) {
        #pragma unroll
        for (int32_t i = 0; i < LOAD_LEN; i++) {
            int32_t vocab_idx = vocab_base + i * TPB;
            // TODO: optimize branch
            if (vocab_idx < vocab_size) {
                local_vals[i + KPT] = logits[batch_offset + vocab_idx];
                local_idxs[i + KPT] = vocab_idx;
            } else {
                local_vals[i + KPT] = -FLT_MAX;
                local_idxs[i + KPT] = -1;
            }
        }
        __syncthreads();
        BlockRadixSort(sort_storage).SortDescendingBlockedToStriped(local_vals, local_idxs);
    }

    float local_top_p = (top_p == nullptr ? top_p_val : top_p[batch_id]);
    if (local_top_p == 0.f) {
        if (threadIdx.x == 0) {
            output[batch_id] = local_idxs[0];
        }
    } else {
        if (threadIdx.x == 0) {
            reducing_memory[0] = local_vals[0];
        }
        __syncthreads();

        float local_sum = 0.f;
        fp32_t max_val = reducing_memory[0];

        for (int32_t i = 0; i < KPT; i++) {
            if (threadIdx.x + i * TPB < top_k_val) {
                local_vals[i] = exp((local_vals[i] - max_val) / temperature);
                local_sum += local_vals[i];
            }
        }

        local_sum = sample_block_reduce_sum<WPT>(local_sum, reducing_memory);
        local_sum = __fdividef(1.f, local_sum + 1e-6f);

        typedef hipcub::BlockScan<float, TPB> BlockScan;
        __shared__ typename BlockScan::TempStorage scan_storage;
        SamplePrefixOp prefix_op(0);

        const fp32_t top_p_selection = local_top_p * (rnd == nullptr ? rnd_val : rnd[batch_id]);
        fp32_t prob_sum = 0.f;
        int32_t count = 0;
        int32_t select_idx;

        for (int32_t i = 0; i < KPT; i++) {
            int32_t top_idx = threadIdx.x + i * TPB;
            fp32_t prob = (top_idx < top_k_val) ? local_vals[i] * local_sum : 1.f;
            BlockScan(scan_storage).InclusiveSum(prob, prob_sum, prefix_op);
            count = __syncthreads_count((int32_t)(prob_sum >= top_p_selection));
            select_idx = local_idxs[i];
            if (count != 0) {
                break;
            }
        }

        if (threadIdx.x == min(TPB - count, TPB - 1)) {
            output[batch_id] = select_idx;
        }
    }
}


template<int32_t TPB, int32_t VPT, int32_t TILE>
__global__
void flash_sample_top_p_kernel(
    const fp32_t __restrict__ *logits,      // [num_batches, batch_stride]
    const fp32_t *temperatures,             // [num_batches]
    const fp32_t *top_p,                    // [num_batches]
    const fp32_t *rnd,                      // [num_batches]
    const int32_t vocab_size,
    const int32_t batch_stride,
    const fp32_t top_p_val,
    const fp32_t rnd_val,
    fp32_t *sorted_value,                   // [num_batches, padded_vocab_size]
    int32_t *sorted_order,                  // [num_batches, padded_vocab_size]
    int32_t *output)                        // [num_batches])
{
    /*
        这是一个投机取巧版本的 Sample Topp 实现，我想它应该是一个非常快的版本。

        Sample Topp 操作要分成几个部分来完成：

            首先要执行一次排序，由于访存无法被合并，这次排序操作会很慢。

            而后要执行 softmax，这意味着三次访存(分别统计 global_max, global_sum, 以及执行 softmax 计算)

            然后你要对 global_sum 乘以一个 [0, 1] 之间的随机数，去执行 sampling 操作，这里我选择使用"接受拒绝采样"

        万幸的是，我们总是可以假设经过充分训练的模型是收敛的，其输出的词表概率应当是"十分尖锐"的——大部分词应该都没什么出现概率，采样过程可以忽略他们。

        因此，我们不执行完整的排序过程，我们采用 局部排序 + flash attention + 多路赢者树归并 的方式实现这个 kernel

        这种实现下的 Sample Topp 比单独执行一次排序还要快。

        这个 kernel 的性能不是稳定的，概率分布越不均衡它越快。但是如果概率分布是均匀的，这个 Kernel 的性能可能会差。
    */
    /* Radix Sort + Softmax + Sampling */

    // allocate shared mem
    constexpr int32_t WARP_SIZE = 32;
    constexpr int32_t WPT = TPB / WARP_SIZE; // warp per thread block.
    typedef hipcub::BlockRadixSort<fp32_t, TPB, VPT, int32_t> BlockRadixSort;
    __shared__ typename BlockRadixSort::TempStorage temp_storage;

    __shared__ fp32_t tile_softmax_m[TPB];
    __shared__ fp32_t tile_softmax_l[TPB];
    __shared__ fp32_t reducing_memory[WPT * 2];
    fp32_t sorting_keys[VPT]; int32_t sorting_values[VPT];

    tile_softmax_m[threadIdx.x] = 0.0f;
    tile_softmax_l[threadIdx.x] = -FLT_MAX;
    __syncthreads();

    // Stage 1. Block Internal Sort & Reducing Max
    const int64_t batch_id            = blockIdx.x;
    const int64_t batch_offset        = batch_id * batch_stride;
    const int64_t padded_batch_offset = batch_id * pad_vocab<fp32_t, TPB>(vocab_size);
    const fp32_t  temperature         = temperatures ? max(abs(temperatures[batch_id]) + 1e-7, 0.01): 1.0f; // temperature 最低 0.01

    for(int32_t block_base_idx = 0; block_base_idx < vocab_size; block_base_idx += TPB * VPT){
        fp32_t local_max = -FLT_MAX, local_sum = 0.0;
        // process multiple elements at once
        const int32_t thread_local_idx = block_base_idx + threadIdx.x;

        # pragma unroll
        for(int32_t internal_loop_idx = 0; internal_loop_idx < VPT; internal_loop_idx++) {
            const int32_t vocab_idx = thread_local_idx + internal_loop_idx * TPB;
            const int64_t load_idx  = batch_offset + vocab_idx;

            if(vocab_idx < vocab_size) {
                const fp32_t value = logits[load_idx] / temperature;
                sorting_keys[internal_loop_idx]   = - value;      // 倒序排序
                sorting_values[internal_loop_idx] = vocab_idx;
                local_max = local_max > value ? local_max : value;
            } else {
                sorting_keys[internal_loop_idx]   = FLT_MAX;
                sorting_values[internal_loop_idx] = -1;
            }
        }

        // Block Reduce max
        __syncthreads();
        local_max = sample_block_reduce_max<WPT>(local_max, reducing_memory);

        // calling Cub::RaidxSort, 这个东西耗时最多，我也不知道咋优化他
        __syncthreads();
        BlockRadixSort(temp_storage).Sort(sorting_keys, sorting_values);

        // 将每一个 block 中的数据写回内存, 此处数据的写回必须合并访存，否则性能很烂
        const int64_t vocab_idx = block_base_idx + threadIdx.x * VPT;
        const int64_t write_idx = vocab_idx + padded_batch_offset;

        # pragma unroll
        for(int32_t internal_loop_idx = 0; internal_loop_idx < VPT; internal_loop_idx++){
            sorting_keys[internal_loop_idx] = exp(- sorting_keys[internal_loop_idx] - local_max);
            local_sum += sorting_keys[internal_loop_idx]; // softmax in this tile
        }
        copy<VPT * sizeof(fp32_t)>(sorting_keys,    &sorted_value[write_idx]);
        copy<VPT * sizeof(int32_t)>(sorting_values, &sorted_order[write_idx]);

        local_sum = sample_block_reduce_sum<WPT>(local_sum, reducing_memory);

        // write block softmax result to shared memory
        // following logic is inspired by flash-attention
        if (threadIdx.x == 0){
            const int32_t tile_idx = block_base_idx / (TPB * VPT);
            tile_softmax_l[tile_idx] = local_max;
            tile_softmax_m[tile_idx] = local_sum;
        }
    }

    fp32_t global_sum = 0.0f;
    fp32_t global_max = -FLT_MAX;
    // flash-attention reduce max, reduce sum
    global_max = sample_block_reduce_max<WPT>(tile_softmax_l[threadIdx.x], reducing_memory);
    __syncthreads();

    global_sum = sample_block_reduce_sum<WPT>(
        tile_softmax_m[threadIdx.x] * exp(tile_softmax_l[threadIdx.x] - global_max),
        reducing_memory);

    fp32_t top_p_selection_rnd = global_sum * (top_p == nullptr ? top_p_val : top_p[batch_id]) * (rnd == nullptr ? rnd_val : rnd[batch_id]);

    // multi way merge-sort & sample top_p
    // 后面这里的采样过程可以进一步优化，但是好像正常来讲不会采样非常多次
    // 需要注意，这是接受拒绝采样，且进行随机访存，如果迟迟无法结束采样，这个 kernel 性能会很差
    // 届时此处的优化将至关重要
    int32_t *selection_slot = reinterpret_cast<int32_t*>(tile_softmax_m);
    selection_slot[threadIdx.x] = 0;
    __syncthreads();

    int64_t base_selection_idx = threadIdx.x * VPT * TPB;
    const fp32_t scale_factor = exp(tile_softmax_l[threadIdx.x] - global_max);

    auto _local_storage = sorting_keys; // 复用一下
    CachedVocabStorage<TPB, VPT> cached_store(
        sorted_value, _local_storage, padded_batch_offset + base_selection_idx);

    for (int32_t selected = 0; selected < vocab_size; selected++) {
        fp32_t selecting_value = -FLT_MAX;
        int32_t select_thread_idx = 0;
        int32_t local_selection = selection_slot[threadIdx.x];

        if (base_selection_idx + local_selection < vocab_size && local_selection < VPT * TPB){
            selecting_value = cached_store.Pop(local_selection);
            selecting_value = selecting_value * scale_factor; // flash softmax
        }

        // block mergesort
        SortingPair p = sample_block_reduce_max_with_index<WPT>(selecting_value, threadIdx.x, reducing_memory);
        select_thread_idx = p.index;
        selecting_value = p.value;
        top_p_selection_rnd -= selecting_value;

        if (threadIdx.x == 0) {
            if (top_p_selection_rnd <= 0.0f) {
                // sampling success, write to output.
                output[batch_id] = sorted_order[
                    padded_batch_offset +
                    select_thread_idx * VPT * TPB +
                    selection_slot[select_thread_idx]
                ];
            }
            selection_slot[select_thread_idx] ++;
        }
        __syncthreads();
        if (top_p_selection_rnd <= 0.0f)
            return;
    }
}

template<int32_t TPB>
__global__
void sample_argmax_kernel(
    const fp32_t* __restrict__ logits, // [batch, batch_stride]
    const int32_t vocab_size,
    const int32_t batch_stride,
    int32_t* output)                   // [batch, 1]
{
    const int64_t batch_id = blockIdx.x;
    int32_t selection_idx = 0;
    fp32_t selecting_value = -FLT_MAX;

    for(int32_t idx = threadIdx.x; idx < vocab_size; idx += TPB) {
        // fp32_t loading = __half2float(logits[batch_id * vocab_size + idx]);
        fp32_t loading = logits[batch_id * batch_stride + idx];
        if (loading > selecting_value) {
            selecting_value = loading;
            selection_idx   = idx;
        }
    }

    // initilize shared memory
    constexpr int32_t WARP_SIZE = 32;
    constexpr int32_t WPT = TPB / WARP_SIZE;
    __shared__ int32_t buffer[TPB];
    __shared__ fp32_t red_smem[WPT * 2];
    buffer[threadIdx.x] = selection_idx;
    __syncthreads();

    SortingPair p = sample_block_reduce_max_with_index<WPT>(selecting_value, threadIdx.x, red_smem);
    if (threadIdx.x == 0)
        output[batch_id] = buffer[p.index];
}

int64_t flash_sample_top_p_get_workspace_size(
    int32_t batch,
    int32_t vocab_size)
{
    return int64_t(batch) * pad_vocab<fp32_t, 256>(vocab_size) * (sizeof(float) + sizeof(int32_t));
}

ppl::common::RetCode flash_sample_topp(
    hipStream_t stream,
    const float* logits, // (batch, batch_stride)
    const float* temperatures, // (batch)
    const float* top_p, // (batch)
    const float* rnd, // (batch)
    const int32_t num_batches,
    const int32_t vocab_size,
    const int32_t batch_stride,
    const float top_p_val,
    const float rnd_val,
    void *workspace,
    int32_t* output) // (batch)
{
    /* Flash Sample Topp

    The FlashSampleTopp function is a high-performance Topp sampling implementation that integrates the functionalities of
        Sorting, Softmax, and Topp sampling.

     sorted_value  {num_batches, pad_vocab<fp32_t, 256>(vocab_size)}

     sorted_index = {num_batches, pad_vocab<fp32_t, 256>(vocab_size)}

    */

    float* sorted_value = (float*)workspace; // (batch, padded_vocab_size)
    int32_t* sorted_index = (int32_t*)sorted_value + pad_vocab<fp32_t, 256>(vocab_size) * sizeof(float); // (batch, padded_vocab_size)

    if (vocab_size <= 32768) {
        flash_sample_top_p_kernel<256, 4, 32>
        <<<num_batches, 256, 0, stream>>>(
            logits, temperatures, top_p,
            rnd, vocab_size, batch_stride,
            top_p_val, rnd_val,
            sorted_value,
            sorted_index,
            output
        );
    } else if (vocab_size <= 262144) {
        flash_sample_top_p_kernel<256, 4, 256>
        <<<num_batches, 256, 0, stream>>>(
            logits, temperatures, top_p,
            rnd, vocab_size, batch_stride,
            top_p_val, rnd_val,
            sorted_value,
            sorted_index,
            output
        );
    } else {
        LOG(ERROR) << "only supporte vocab_size <= 262144, vocab_size = " << vocab_size;
        return ppl::common::RC_UNSUPPORTED;
    }

    return ppl::common::RC_SUCCESS;
}

int64_t sample_topk_topp_get_workspace_size(int32_t batch, int32_t vocab_size, int32_t top_k_val) {
    int64_t buffer_size = 0;
    if (top_k_val != 1 && top_k_val <= 12) {
        buffer_size = int64_t(batch) * pad_vocab<fp32_t, 256>(vocab_size) * sizeof(float);
    }
    return buffer_size;
}

ppl::common::RetCode sample_topk_topp(
    hipStream_t stream,
    const float* logits, // (batch, batch_stride)
    const float* temperatures, // (batch)
    const float* top_p, // (batch)
    const float* rnd, // (batch)
    const int32_t num_batches,
    const int32_t vocab_size,
    const int32_t batch_stride,
    const int32_t top_k_val,
    const float top_p_val,
    const float rnd_val,
    void *workspace,
    int32_t* output) // (batch)
{
    constexpr int32_t TPB = 256;
    constexpr int32_t VPT = 4;
    constexpr int32_t LPT = 32;     // sort length per thread

    float* padded_logits = (float*)workspace; // (batch, padded_vocab_szie)

    if (top_k_val == 1 || (!top_p && top_p_val == 0.0f)) {
        sample_argmax_kernel<TPB>
        <<<num_batches, TPB, 0, stream>>>(
            logits, vocab_size, batch_stride, output
        );
    } else if (top_k_val <= 12) {
        sample_topk_topp_default_kernel<TPB, VPT>
        <<<num_batches, TPB, top_k_val * 8, stream>>>(
            logits, temperatures, top_p, rnd,
            vocab_size, batch_stride,
            top_k_val, top_p_val, rnd_val,
            padded_logits, output
        );
    } else if (top_k_val <= 256) {
        sample_topk_topp_radix_select_kernel<TPB, LPT, 1>
        <<<num_batches, TPB, 0, stream>>>(
            logits, temperatures, top_p, rnd,
            vocab_size, batch_stride,
            top_k_val, top_p_val, rnd_val,
            output
        );
    } else if (top_k_val <= 512) {
        sample_topk_topp_radix_select_kernel<TPB, LPT, 2>
        <<<num_batches, TPB, 0, stream>>>(
            logits, temperatures, top_p, rnd,
            vocab_size, batch_stride,
            top_k_val, top_p_val, rnd_val,
            output
        );
    } else if (top_k_val <= 768) {
        sample_topk_topp_radix_select_kernel<TPB, LPT, 3>
        <<<num_batches, TPB, 0, stream>>>(
            logits, temperatures, top_p, rnd,
            vocab_size, batch_stride,
            top_k_val, top_p_val, rnd_val,
            output
        );
    } else if (top_k_val <= 1024) {
        sample_topk_topp_radix_select_kernel<TPB, LPT, 4>
        <<<num_batches, TPB, 0, stream>>>(
            logits, temperatures, top_p, rnd,
            vocab_size, batch_stride,
            top_k_val, top_p_val, rnd_val,
            output
        );
    } else {
        LOG(ERROR) << "only supporte top_k <= 1024, top_k = " << top_k_val;
        return ppl::common::RC_UNSUPPORTED;
    }

    return ppl::common::RC_SUCCESS;
}

ppl::common::RetCode sample_argmax(
    hipStream_t stream,
    const float* logits, // (batch, batch_stride)
    const int32_t num_batches,
    const int32_t vocab_size,
    const int32_t batch_stride,
    int32_t* output) // (batch)
{
    sample_argmax_kernel<256><<<num_batches, 256, 0, stream>>>(logits, vocab_size, batch_stride, output);

    return ppl::common::RC_SUCCESS;
}

}}}}}
