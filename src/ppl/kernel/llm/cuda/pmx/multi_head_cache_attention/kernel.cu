#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include "ppl/kernel/llm/cuda/pmx/multi_head_cache_attention.h"
#include "ppl/common/log.h"
#include "../type.h"

#include "ppl/kernel/llm/cuda/xformer/fmha.h"
#include "ppl/kernel/llm/cuda/flash_attn2/fmha.h"
#include "cudakernel/common/common.cuh"

#include <hip/hip_fp16.h>
#include <float.h> // need for FLT_MAX

namespace ppl { namespace kernel { namespace llm { namespace cuda { namespace pmx {

static constexpr int32_t UNIFORM_PAGE_SIZE = 128;

using decoding_algo = dynamic_batching_multi_head_cache_attention::decoding_algo;

struct dynamic_batching_decoding_cache_attention_kernel_param {
    half* query;
    half* attn_mask;
    half* output;
    int8_t* cache;
    half* scale;
    int64_t* cachestarts;
    int64_t* kvstarts;
    float attn_scale;
    int64_t layer_idx;
    int64_t num_kv_repeats;
    int64_t page_size;
    int64_t query_stride_s;
    int64_t output_stride_s;
    int64_t mask_stride_s;
    int64_t mask_stride_h;
    int64_t cache_stride_s;
    int64_t cache_stride_l;
    int64_t cache_stride_h;
    int64_t cache_stride_kv;
    int64_t cachestarts_stride_b;

    struct {
        int32_t* block_counter;
        float* partial_log_sum_exp;
        half* partial_out;
    } multi_block;
};

struct dynamic_batching_kv_cache_quantize_kernel_param {
    half* current_key; // (S, KVH..., D)
    half* current_value; // (S, KVH..., D)
    int64_t* seqstarts; // (B + 1)
    int64_t* cachestarts;// (B) or (B, MaxP)
    int64_t* start_pos; // (B)
    int64_t num_layer;
    int64_t layer_idx;
    int64_t num_kv_heads;
    int64_t head_dim;
    int32_t page_size;
    int64_t current_key_stride_s;
    int64_t current_value_stride_s;
    int64_t cache_stride_s;
    int64_t cache_stride_l;
    int64_t cache_stride_h;
    int64_t cache_stride_kv;
    int64_t cachestarts_stride_b;
    int8_t* cache;
    half* scale;
};

template<int32_t VPT, int32_t TPB, int32_t PAGE_SIZE> // 8 fp16 occupy 128 bytes, which can be loaded by a single thread at once.
__global__
void dynamic_batching_kv_cache_quantize_kernel(dynamic_batching_kv_cache_quantize_kernel_param p)
{
    if (blockIdx.x < p.seqstarts[blockIdx.y + 1] - p.seqstarts[blockIdx.y]) {
        const int64_t thr_per_head = p.head_dim / VPT;
        const int64_t batch_id = blockIdx.y;
        const int64_t seq_idx = blockIdx.x;
        const int64_t tid = blockIdx.z * TPB + threadIdx.x;

        if (tid < p.num_kv_heads * p.head_dim / VPT) {
            const int64_t cache_token_idx = PAGE_SIZE <= 0
                ? (p.cachestarts[batch_id] + seq_idx + p.start_pos[batch_id])
                : (p.cachestarts[batch_id * p.cachestarts_stride_b + (seq_idx + p.start_pos[batch_id]) / PAGE_SIZE]
                    + ((seq_idx + p.start_pos[batch_id]) % PAGE_SIZE));
            const int64_t input_token_idx = p.seqstarts[batch_id] + seq_idx;
            const int64_t key_out_offset = cache_token_idx * p.cache_stride_s + p.layer_idx * p.cache_stride_l;
            auto key_in_ptr = p.current_key + input_token_idx * p.current_key_stride_s;
            auto value_in_ptr = p.current_value + input_token_idx * p.current_value_stride_s;

            const int64_t kv_head_idx = tid / (thr_per_head);
            const int64_t dim_idx = (tid % thr_per_head) * VPT;
            const int64_t scale_dim_idx = dim_idx / VPT;
            const int64_t input_idx = kv_head_idx * p.head_dim + dim_idx;

            half key_in[VPT]; int8_t key_out[VPT];
            half value_in[VPT]; int8_t value_out[VPT];

            copy<sizeof(half) * VPT>(&key_in_ptr[input_idx], key_in);
            copy<sizeof(half) * VPT>(&value_in_ptr[input_idx], value_in);

            const int64_t key_out_idx
                = key_out_offset
                + kv_head_idx * p.cache_stride_h
                + dim_idx;
            const int64_t value_out_idx
                = key_out_idx
                + p.cache_stride_kv;

            const int64_t key_scale_out_idx = (key_out_idx - dim_idx) / VPT + scale_dim_idx;
            const int64_t value_scale_out_idx = key_scale_out_idx + p.cache_stride_kv / VPT;

            // calculate kv scale
            const half eps = 1e-5f;
            half key_max = 0.0f;
            half value_max = 0.0f;

            #pragma unroll
            for (int32_t i = 0; i < VPT; i ++){
                key_max = key_max > __habs(key_in[i]) ? key_max : __habs(key_in[i]);
                value_max = value_max > __habs(value_in[i]) ? value_max : __habs(value_in[i]);
            }

            half key_scale = __float2half(__half2float(key_max) / 127.0f);
            half value_scale = __float2half(__half2float(value_max) / 127.0f);
            key_scale = key_scale > eps ? key_scale : eps;
            value_scale = value_scale > eps ? value_scale : eps;

            #pragma unroll
            for (int32_t i = 0; i < VPT; i ++){
                key_out[i] = (int8_t)__half2short_rn(key_in[i] / key_scale);
                value_out[i] = (int8_t)__half2short_rn(value_in[i] / value_scale);
            }

            copy<sizeof(int8_t) * VPT>(key_out, &p.cache[key_out_idx]);
            copy<sizeof(int8_t) * VPT>(value_out, &p.cache[value_out_idx]);

            p.scale[key_scale_out_idx] = key_scale;
            p.scale[value_scale_out_idx] = value_scale;
        }
    }
}

template<int32_t THREAD_GROUP_SIZE>
__device__ inline
float attn_thread_group_reduce_sum(float qk)
{
#pragma unroll
    for (int32_t mask = THREAD_GROUP_SIZE / 2; mask >= 1; mask /= 2) {
        qk += __shfl_xor_sync(uint32_t(-1), qk, mask);
    }
    return qk;
}

template<int32_t WPT, int32_t STOP_MASK>
__device__ inline
float attn_block_reduce_max(float reducing, float* shared_mem)
{
    // Helper function for reduce softmax qkmax.
    constexpr int32_t WARP_SIZE = 32;
    const int32_t lane_id = threadIdx.x % WARP_SIZE;
    const int32_t warp_id = threadIdx.x / WARP_SIZE;

# pragma unroll
    for (int32_t mask = WARP_SIZE / 2; mask >= STOP_MASK; mask /= 2) {
        reducing = fmaxf(reducing, __shfl_xor_sync(uint32_t(-1), reducing, mask));
    }

    if (lane_id == 0) {
        shared_mem[warp_id] = reducing;
    }
    __syncthreads();

    if (lane_id < WPT) reducing = shared_mem[lane_id];
    else reducing = -FLT_MAX;

# pragma unroll
    for (int32_t mask = WPT / 2; mask >= 1; mask /= 2) {
        reducing = fmaxf(reducing, __shfl_xor_sync(uint32_t(-1), reducing, mask));
    }

    reducing = __shfl_sync(uint32_t(-1), reducing, 0);
    return reducing;
}

template<int32_t WPT, int32_t STOP_MASK>
__device__ inline
float attn_block_reduce_sum(float reducing, float *shared_mem)
{
    // Helper function for reduce softmax exp sum.
    constexpr int32_t WARP_SIZE = 32;
    const int32_t lane_id = threadIdx.x % WARP_SIZE;
    const int32_t warp_id = threadIdx.x / WARP_SIZE;

# pragma unroll
    for (int32_t mask = WARP_SIZE / 2; mask >= STOP_MASK; mask /= 2) {
        reducing += __shfl_xor_sync(uint32_t(-1), reducing, mask);
    }

    if (lane_id == 0) shared_mem[warp_id] = reducing;
    __syncthreads();

    if (lane_id < WPT) reducing = shared_mem[lane_id];

# pragma unroll
    for (int32_t mask = WPT / 2; mask >= 1; mask /= 2) {
        reducing += __shfl_xor_sync(uint32_t(-1), reducing, mask);
    }
    reducing = __shfl_sync(uint32_t(-1), reducing, 0);
    return reducing;
}


template<
    int32_t HEAD_SIZE,          // head dimension
    int32_t THREAD_GROUP_SIZE,  // how many threads inside a group (each group deal with one context)
    int32_t TPB,                // threads per block
    int32_t QUANT_GROUP,
    int32_t MULTI_BLOCK,        // do flash decoding if more than 1
    bool    ATTN_MASK,
    int32_t PAGE_SIZE>
__global__
void dynamic_batching_decoding_cache_sharemem_attention_fp16_kernel(dynamic_batching_decoding_cache_attention_kernel_param p)
{
    /***
    * You have to remember that this Kernel was created by a brother on the night of July 20, 2023. On that day,
    * Beijing experienced the strongest rainstorm since the beginning of summer.        --ZhiLao /doge

    DecodingAttention is a special operator designed specifically for large language models(LLM) decoding.

    It requires that the length of each input Query is always 1,
        while the Key and Value can have different lengths.

    This operator supports padding removal optimization, meaning that Q, K, and V all need to have their tokens
        concentrated in one sentence for input, with shapes like Q: [seq_lens, num_heads, head_size],
        and K: [context_lens, num_kv_heads, head_size].

    Since the Query sentence length is always 1, this operator is literally a fused matrix-vector multiplications operation.
        It does not utilize tensor cores for computation.

    The calculation logic is divided into three steps: gemv(QK) + softmax(Attention) + gemv(KV).
        In the provided code, it has already been split into these three parts.
    ***/

    /* --- Decoding Attention Kernel Implementation --- */

    // magic number for quick convert from int8 to fp16
    static constexpr uint32_t I8s_TO_F16s_MAGIC_NUM = 0x64806480;
    static constexpr uint32_t mask_for_elt_01       = 0x5150;
    static constexpr uint32_t mask_for_elt_23       = 0x5352;
    static constexpr uint32_t start_byte_for_fp16   = 0x64646464;

    constexpr int64_t WARP_SIZE = 32;                                   // warp size
    constexpr int64_t WPT       = TPB / WARP_SIZE;                      // warp per thread block
    constexpr int64_t GPW       = WARP_SIZE / THREAD_GROUP_SIZE;        // thread group per warp
    constexpr int64_t GPT       = WARP_SIZE / THREAD_GROUP_SIZE * WPT;  // thread group per thread block

    // const int64_t num_heads     = gridDim.x;
    const int64_t num_batchs    = gridDim.y;
    const int32_t head_idx      = blockIdx.x;
    const int64_t batch_idx     = blockIdx.y;
    const int64_t block_idx     = blockIdx.z;           // multi-block index for flash decoding
    constexpr int64_t VEC_SIZE  = 16 / sizeof(half);    // num of fp16 inside a 128bit vector for memory loading and storing

    // ------------------------------------------------ //
    // Step 1. Load Q into Thread Reg.
    constexpr int64_t VEC_LEN = (HEAD_SIZE / VEC_SIZE) / THREAD_GROUP_SIZE; // num of vecotr for each thread handles

    static_assert((HEAD_SIZE / THREAD_GROUP_SIZE) % VEC_SIZE == 0);
    static_assert(HEAD_SIZE % THREAD_GROUP_SIZE == 0);
    static_assert(QUANT_GROUP == 8);

    constexpr int64_t QUANT_GROUP_SHIFT = 3;

    // The elements in Q, K, and V will be evenly distributed across each thread group.
    half local_q[VEC_SIZE * VEC_LEN];

    const int64_t warp_id       = threadIdx.x / WARP_SIZE;
    const int64_t warp_lane_id  = threadIdx.x % WARP_SIZE;
    const int64_t group_id      = warp_lane_id / THREAD_GROUP_SIZE;
    const int64_t group_lane_id = warp_lane_id % THREAD_GROUP_SIZE;

    const int64_t cache_offset_s  = PAGE_SIZE <= 0 ? p.cachestarts[batch_idx] : 0;  // base address of cache for each batch
    const int32_t kv_head_idx     = head_idx / int32_t(p.num_kv_repeats);           // same of head_idx if not using GQA

    fp16_t *partial_o       = nullptr;  // base address for each head to store partial output generated by flash decoding
    fp32_t *partial_log_sum = nullptr;  // partial log sum exp for flash decoding
    int32_t *block_counter  = nullptr;  // block counter for flash decoding to select the final block to do final reduction
    if (MULTI_BLOCK > 1) {
        partial_o
            = p.multi_block.partial_out
            + batch_idx * HEAD_SIZE * MULTI_BLOCK
            + head_idx * num_batchs * HEAD_SIZE * MULTI_BLOCK;
        partial_log_sum
            = p.multi_block.partial_log_sum_exp
            + batch_idx * MULTI_BLOCK
            + head_idx * num_batchs * MULTI_BLOCK;
        block_counter
            = p.multi_block.block_counter
            + batch_idx
            + head_idx * num_batchs;
    }

    half *attn_mask = nullptr;
    if (ATTN_MASK) {
        attn_mask = p.attn_mask
                + p.mask_stride_h * head_idx
                + batch_idx * p.mask_stride_s
                + p.kvstarts[batch_idx];
    }

    // load Q from global memory to registers by vectorized accesss(128 bit)
    // every THREAD_GROUP load the same head of Q for one block
    #pragma unroll
    for (int64_t i = 0; i < VEC_LEN; i++) {
        copy<sizeof(half) * VEC_SIZE>(
            &p.query[
                batch_idx * p.query_stride_s +
                head_idx * HEAD_SIZE +
                (group_lane_id + i * THREAD_GROUP_SIZE) * VEC_SIZE
            ],
            &local_q[i * VEC_SIZE]);
    }
    // ------------------------------------------------ //
    // Step 2. Solve QK Dot

    // In the process of handling the QK matrix multiplication, we will divide a complete Thread Warp into several Thread groups.
    // Each thread group reads the entire Query and saves it in registers.
    // Then, each thread group iterates through the vectors in the Key and performs dot products with the Query.
    // During this process, a WARP performs multiple vector dot product operations at once.
    // At the same time, we also record the maximum current_value of the dot product results for later use in the softmax operation.
    const int64_t context_len           = p.kvstarts[batch_idx + 1] - p.kvstarts[batch_idx];    // input context len
    const int64_t context_len_per_block = (context_len + MULTI_BLOCK - 1) / MULTI_BLOCK;        // context len for each multi-block but not the last one
    const int64_t block_context_beg     = block_idx * context_len_per_block;                    // base context index for each multi-block
    // set the valid context len for every multi-block with the last one
    const int64_t block_context_len     = context_len >= context_len_per_block * (block_idx + 1) ? context_len_per_block : context_len - block_context_beg;

    extern __shared__ float logits[];
    float partial_qk_max = -FLT_MAX;

    for (int64_t base_id = warp_id * GPW; base_id < block_context_len; base_id += GPT) {
        int8_t local_k_quant[VEC_SIZE * VEC_LEN];
        half local_k_scale[VEC_LEN];
        const int64_t block_context_id = base_id + group_id;

        float qk_dot = 0.0f;

        // all thread groups within a warp must be launched together.
        if (block_context_id < block_context_len) {
            const int64_t cache_token_idx = PAGE_SIZE <= 0
                            ? (cache_offset_s + block_context_beg + block_context_id)
                            : (p.cachestarts[batch_idx * p.cachestarts_stride_b + (block_context_beg + block_context_id) / PAGE_SIZE]
                                + ((block_context_beg + block_context_id) % PAGE_SIZE));
            const int64_t key_offset
                            = cache_token_idx * p.cache_stride_s
                            + p.layer_idx * p.cache_stride_l
                            + p.cache_stride_h * kv_head_idx
                            + group_lane_id * VEC_SIZE;
            #pragma unroll
            for (int64_t i = 0; i < VEC_LEN; i++) {
                // copy 128(16 * 8) bits from K to Local K
                const int64_t key_idx = key_offset + i * THREAD_GROUP_SIZE * VEC_SIZE;
                // load int8-K from kvcache to registers
                copy<sizeof(int8_t) * VEC_SIZE>(&p.cache[key_idx],  &local_k_quant[i * VEC_SIZE]);
                const int64_t key_scale_idx = key_idx >> QUANT_GROUP_SHIFT;
                local_k_scale[i] = p.scale[key_scale_idx];

                // fast convert from int8 to fp16
                #pragma unroll
                for(int64_t k = 0; k < VEC_SIZE; k++) {
                    local_k_quant[i * VEC_SIZE + k] += 128;
                }
                half result[8];
                uint32_t*      h   = reinterpret_cast<uint32_t*>(result);
                uint32_t const i8s = reinterpret_cast<uint32_t const&>(*(local_k_quant + i * VEC_SIZE));
                asm volatile("prmt.b32 %0,%1,%2,%3;\n" : "=r"(h[0]) : "r"(i8s), "n"(start_byte_for_fp16), "n"(mask_for_elt_01));
                asm volatile("prmt.b32 %0,%1,%2,%3;\n" : "=r"(h[1]) : "r"(i8s), "n"(start_byte_for_fp16), "n"(mask_for_elt_23));
                asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h[0]) : "r"(h[0]), "r"(I8s_TO_F16s_MAGIC_NUM));
                asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h[1]) : "r"(h[1]), "r"(I8s_TO_F16s_MAGIC_NUM));
                uint32_t*      h_2   = reinterpret_cast<uint32_t*>(result+4);
                uint32_t const i8s_2 = reinterpret_cast<uint32_t const&>(*(local_k_quant + i * VEC_SIZE + 4));
                asm volatile("prmt.b32 %0,%1,%2,%3;\n" : "=r"(h_2[0]) : "r"(i8s_2), "n"(start_byte_for_fp16), "n"(mask_for_elt_01));
                asm volatile("prmt.b32 %0,%1,%2,%3;\n" : "=r"(h_2[1]) : "r"(i8s_2), "n"(start_byte_for_fp16), "n"(mask_for_elt_23));
                asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h_2[0]) : "r"(h_2[0]), "r"(I8s_TO_F16s_MAGIC_NUM));
                asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h_2[1]) : "r"(h_2[1]), "r"(I8s_TO_F16s_MAGIC_NUM));
                // compute partial qk-dot in one context for each thread
                #pragma unroll
                for (int64_t j = 0; j < VEC_SIZE; j++) {
                    qk_dot += __half2float(local_q[i * VEC_SIZE + j]) * __half2float(local_k_scale[i] * result[j]);
                }
            }
        }

        // every thread group get a full qk-dot in one context
        qk_dot = p.attn_scale * attn_thread_group_reduce_sum<THREAD_GROUP_SIZE>(qk_dot);

        // save qk-dot for each context and update max qk-dot
        if (group_lane_id == 0 && block_context_id < block_context_len) {
            if (ATTN_MASK)
                qk_dot += __half2float(attn_mask[block_context_beg + block_context_id]);
            logits[block_context_id] = qk_dot;
            partial_qk_max = fmaxf(qk_dot, partial_qk_max);
       }
    }

    // ------------------------------------------------ //
    // Step 3. Softmax

    // The process of solving softmax is divided into two stages.
    // First, we need to reduce partial_qk_max in two dimensions: WARP and ThreadBlock.
    // Afterward, we use reduced partial_qk_max to perform softmax calculations,
    //    the results will all be stored in shared memory.
    __shared__ float red_smem[WPT];

    // reduce partial_qk_max in thread block and boardcast
    partial_qk_max = attn_block_reduce_max<WPT, 1>(partial_qk_max, red_smem);

    // Softmax Kernel Logic Start here
    // convert qk-dot to exp(local-qk-dot - max-qk-dot) in shared memory
    // sum up all exp(local-qk-dot - max-qk-dot)
    float partial_exp_sum = 0.0f;
    for (int64_t block_context_id = threadIdx.x; block_context_id < block_context_len; block_context_id += TPB){
        logits[block_context_id] -= partial_qk_max;
        logits[block_context_id] = exp(logits[block_context_id]);
        partial_exp_sum += logits[block_context_id];
    }

    // block reduce sum on partial_exp_sum
    // Warp per thread block must be power-of-2 for reducation, check attn_block_reduce_sum kernel.
    static_assert(WPT == 2 || WPT == 4 || WPT == 8 || WPT == 16 || WPT == 32 || WPT == 64);
    partial_exp_sum = attn_block_reduce_sum<WPT, 1>(partial_exp_sum, red_smem);

    // save partial log sum exp for flash decoding
    if (MULTI_BLOCK > 1 && threadIdx.x == 0) {
        partial_log_sum[block_idx] = partial_qk_max + log(partial_exp_sum);
    }

    // ------------------------------------------------ //
    // Step 4. Solve logits * V

    int8_t local_v_quant[VEC_SIZE * VEC_LEN];
    float local_v[VEC_SIZE * VEC_LEN];
    half local_v_scale[VEC_LEN];

    #pragma unroll
    for(int32_t i = 0; i < VEC_SIZE * VEC_LEN; i++) {
        local_v[i] = 0;
    }

    for (int64_t base_id = warp_id * GPW; base_id < block_context_len; base_id += GPT) {
        const int64_t block_context_id = base_id + group_id;
        // all thread groups within a warp must be launched together.
        if (block_context_id < block_context_len) {
            const int64_t cache_token_idx = PAGE_SIZE <= 0
                            ? (cache_offset_s + block_context_beg + block_context_id)
                            : (p.cachestarts[batch_idx * p.cachestarts_stride_b + (block_context_beg + block_context_id) / PAGE_SIZE]
                                + ((block_context_beg + block_context_id) % PAGE_SIZE));
            const int64_t value_offset
                            = cache_token_idx * p.cache_stride_s
                            + p.layer_idx * p.cache_stride_l
                            + p.cache_stride_h * kv_head_idx
                            + group_lane_id * VEC_SIZE
                            + p.cache_stride_kv;
            #pragma unroll
            for (int64_t i = 0; i < VEC_LEN; i++) {
                // copy 128(16 * 8) bits from V to Local V
                const int64_t value_idx = value_offset + i * THREAD_GROUP_SIZE * VEC_SIZE;
                // load int8-V from kvcache to registers
                copy<sizeof(int8_t) * VEC_SIZE>(&p.cache[value_idx],  &local_v_quant[i * VEC_SIZE]);
                const int64_t value_scale_idx = value_idx >> QUANT_GROUP_SHIFT;
                local_v_scale[i] = p.scale[value_scale_idx];

                // fast convert from int8 to fp16
                #pragma unroll
                for(int64_t k = 0; k < VEC_SIZE; k++) {
                    local_v_quant[i * VEC_SIZE + k] += 128;
                }
                half result[8];
                uint32_t*      h   = reinterpret_cast<uint32_t*>(result);
                uint32_t const i8s = reinterpret_cast<uint32_t const&>(*(local_v_quant + i * VEC_SIZE));
                asm volatile("prmt.b32 %0,%1,%2,%3;\n" : "=r"(h[0]) : "r"(i8s), "n"(start_byte_for_fp16), "n"(mask_for_elt_01));
                asm volatile("prmt.b32 %0,%1,%2,%3;\n" : "=r"(h[1]) : "r"(i8s), "n"(start_byte_for_fp16), "n"(mask_for_elt_23));
                asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h[0]) : "r"(h[0]), "r"(I8s_TO_F16s_MAGIC_NUM));
                asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h[1]) : "r"(h[1]), "r"(I8s_TO_F16s_MAGIC_NUM));
                uint32_t*      h_2   = reinterpret_cast<uint32_t*>(result+4);
                uint32_t const i8s_2 = reinterpret_cast<uint32_t const&>(*(local_v_quant + i * VEC_SIZE + 4));
                asm volatile("prmt.b32 %0,%1,%2,%3;\n" : "=r"(h_2[0]) : "r"(i8s_2), "n"(start_byte_for_fp16), "n"(mask_for_elt_01));
                asm volatile("prmt.b32 %0,%1,%2,%3;\n" : "=r"(h_2[1]) : "r"(i8s_2), "n"(start_byte_for_fp16), "n"(mask_for_elt_23));
                asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h_2[0]) : "r"(h_2[0]), "r"(I8s_TO_F16s_MAGIC_NUM));
                asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h_2[1]) : "r"(h_2[1]), "r"(I8s_TO_F16s_MAGIC_NUM));
                // v * sum(exp(context_qk_dot - max_qk_dot))
                #pragma unroll
                for (int64_t j = 0; j < VEC_SIZE; j++) {
                    local_v[i * VEC_SIZE + j] += __half2float(local_v_scale[i] * result[j]) * logits[block_context_id];
                }
            }
        }
    }

    // complete softmax in local_v by dividing partial_exp_sum to generate partial output in local_v
    const float inv_sum = __fdividef(1.f, partial_exp_sum + 1e-6f);
    #pragma unroll
    for (int32_t i = 0; i < VEC_SIZE * VEC_LEN; i++) {
        local_v[i] *= inv_sum;
        #pragma unroll
        for (int32_t mask = THREAD_GROUP_SIZE; mask <= WARP_SIZE >> 1; mask = mask << 1) {
            local_v[i] += __shfl_xor_sync(uint32_t(-1), local_v[i], mask);
        }
    }
    //for now, every warp's each thread group got the partial result inside a warp
    //we need to add up each warp's first thread group by reusing the logits smem

    // wait for logits to be reused
    __syncthreads();

    constexpr int64_t WORK_THREAD = WPT * THREAD_GROUP_SIZE * VEC_LEN;          // num of thread needed to complete block output reduction
    constexpr int64_t WORK_WARP = (WORK_THREAD + WARP_SIZE - 1) / WARP_SIZE;    // num of warp needed for reduction
    constexpr int64_t VPT = 16;                     // 16 * 8bit
    constexpr int64_t V32PT = 16 / sizeof(float);   // num of fp32 inside a vector

    const int32_t v_warp_id  = threadIdx.x % WPT;                           // warp index of reduce data for each thread to load
    const int32_t v_group_id = (threadIdx.x / WPT) % THREAD_GROUP_SIZE;     // group index of reduce data for each thread to load
    const int32_t v_vec_id   = threadIdx.x / (WPT * THREAD_GROUP_SIZE);     // vector index of reduce data for each thread to load

    half local_out[VEC_SIZE];

    // save local_v to shared memory without bank conflict
    if (warp_lane_id < THREAD_GROUP_SIZE) {
        #pragma unroll
        for (int32_t i = 0; i < VEC_LEN * VEC_SIZE; i += V32PT) {
            copy<VPT>(
                &local_v[i],
                &logits[
                    i * WPT * THREAD_GROUP_SIZE +
                    warp_lane_id * WPT * V32PT +
                    ((warp_id + warp_lane_id) % WPT) * V32PT]);
        }
    }

    __syncthreads();

    // WPT reduce
    if (warp_id < WORK_WARP) {
        // each thread only load VEC_SIZE of partial ouput
        if (threadIdx.x < WORK_THREAD) {
            #pragma unroll
            for (int32_t i = 0; i < VEC_SIZE; i+= V32PT) {
                copy<VPT>(
                    &logits[
                        v_vec_id * VEC_SIZE * WPT * THREAD_GROUP_SIZE +
                        i * WPT * THREAD_GROUP_SIZE +
                        v_group_id * WPT * V32PT +
                        ((v_warp_id + v_group_id) % WPT) * V32PT],
                    &local_v[i]);
            }
        } else {
            for (int32_t i = 0; i < VEC_SIZE * VEC_LEN; i+= 1) {
                local_v[i] = 0.f;
            }
        }
        // block reduce sum on ouput
        #pragma unroll
        for (int32_t i = 0; i < VEC_SIZE; i++) {
            #pragma unroll
            for (int32_t mask = WPT / 2; mask >= 1; mask /= 2) {
                local_v[i] += __shfl_xor_sync(uint32_t(-1), local_v[i], mask);
            }
            local_out[i] = __float2half(local_v[i]);
        }
        if (v_warp_id == 0) {
            // save block ouput to final address or buffer for flash decoding
            half* partial_out = (MULTI_BLOCK == 1)
                    ? &p.output[
                        batch_idx * p.output_stride_s +
                        head_idx * HEAD_SIZE +
                        v_vec_id * THREAD_GROUP_SIZE * VEC_SIZE +
                        v_group_id * VEC_SIZE]
                    : &partial_o[
                        (v_vec_id * THREAD_GROUP_SIZE + v_group_id) * MULTI_BLOCK * VEC_SIZE
                        + block_idx * VEC_SIZE];
            copy<VPT>(local_out, partial_out);
        }
    }

    // Flash decoding
    if (MULTI_BLOCK > 1) {
        __syncthreads();

        bool last_block = false;
        // Make sure every block finishs the partial computation.
        if (threadIdx.x == 0) {
            if (atomicAdd(block_counter, 1) == MULTI_BLOCK - 1) {
                last_block = true;
            }
        }

        // The last block do the final computation.
        if (__syncthreads_or(last_block)) {
            const int64_t multi_block_idx = threadIdx.x % MULTI_BLOCK;

            // get max block log sum exp
            float local_log_sum_exp = warp_lane_id < MULTI_BLOCK ? partial_log_sum[multi_block_idx] : -FLT_MAX;
            float max_log_sum_exp = local_log_sum_exp;
            # pragma unroll
            for (int32_t mask = MULTI_BLOCK / 2; mask >= 1; mask /= 2) {
                max_log_sum_exp = fmaxf(max_log_sum_exp, __shfl_xor_sync(uint32_t(-1), max_log_sum_exp, mask));
            }
            max_log_sum_exp = __shfl_sync(uint32_t(-1), max_log_sum_exp, 0);

            // update scale
            float local_scale = warp_lane_id < MULTI_BLOCK ? exp(local_log_sum_exp - max_log_sum_exp) : 0.f;
            float scale_sum = local_scale;
            # pragma unroll
            for (int32_t mask = MULTI_BLOCK / 2; mask >= 1; mask /= 2) {
                scale_sum += __shfl_xor_sync(uint32_t(-1), scale_sum, mask);
            }
            scale_sum = __shfl_sync(uint32_t(-1), scale_sum, 0);

            float *scale_smem = logits;
            int scale_id = warp_id * MULTI_BLOCK + warp_lane_id;
            if (warp_lane_id < MULTI_BLOCK && scale_id < WARP_SIZE) {
                scale_smem[scale_id] = local_scale / scale_sum;
            }
            __syncthreads();

            // final reduce for multi-block output
            const int64_t head_dim_idx_base   = threadIdx.x / MULTI_BLOCK * VEC_SIZE;
            const int64_t head_dim_idx_stride = TPB / MULTI_BLOCK * VEC_SIZE;

            #pragma unroll
            for (int64_t head_dim_offset = 0; head_dim_offset < HEAD_SIZE; head_dim_offset += head_dim_idx_stride) {
                int64_t head_dim_idx = head_dim_idx_base + head_dim_offset;
                half final_out[VEC_SIZE];
                local_scale = scale_smem[warp_lane_id];
                if (head_dim_idx < HEAD_SIZE) {
                    copy<VEC_SIZE*sizeof(half)>(
                        &partial_o[
                            head_dim_idx * MULTI_BLOCK +
                            multi_block_idx * VEC_SIZE],
                        final_out);
                }

                #pragma unroll
                for (int32_t i = 0; i < VEC_SIZE; i++) {
                    float float_out = __half2float(final_out[i]) * local_scale;
                    # pragma unroll
                    for (int32_t mask = MULTI_BLOCK / 2; mask >= 1; mask /= 2) {
                        float_out += __shfl_xor_sync(uint32_t(-1), float_out, mask);
                    }
                    final_out[i] = __float2half(float_out);
                }

                if (multi_block_idx == 0 && head_dim_idx < HEAD_SIZE) {
                    copy<VPT>(
                        final_out,
                        &p.output[
                            batch_idx * p.output_stride_s +
                            head_idx * HEAD_SIZE +
                            head_dim_idx]);
                }
            }
        }
    }
}


template<
    int32_t HEAD_SIZE,          // head dimension
    int32_t THREAD_GROUP_SIZE,  // how many threads inside a group
    int32_t TPB,                // threads per block
    int32_t QUANT_GROUP,
    int32_t MULTI_BLOCK,        // do flash decoding if more than 1
    bool    ATTN_MASK,
    int32_t PAGE_SIZE>
__global__
void dynamic_batching_decoding_cache_infinity_attention_fp16_kernel(dynamic_batching_decoding_cache_attention_kernel_param p)
{
    static constexpr uint32_t I8s_TO_F16s_MAGIC_NUM = 0x64806480;
    static constexpr uint32_t mask_for_elt_01       = 0x5150;
    static constexpr uint32_t mask_for_elt_23       = 0x5352;
    static constexpr uint32_t start_byte_for_fp16   = 0x64646464;

    constexpr int64_t WARP_SIZE = 32;                              // warp size
    constexpr int64_t WPT       = TPB / WARP_SIZE;                 // warp per thread block
    constexpr int64_t GPW       = WARP_SIZE / THREAD_GROUP_SIZE;       // thread group per warp
    constexpr int64_t GPT       = WARP_SIZE / THREAD_GROUP_SIZE * WPT; // thread group per thread block

    // const int64_t num_heads     = gridDim.x;
    const int64_t num_batchs    = gridDim.y;
    const int32_t head_idx      = blockIdx.x;
    const int64_t batch_idx     = blockIdx.y;
    const int64_t block_idx     = blockIdx.z;
    constexpr int64_t VEC_SIZE  = 16 / sizeof(half);  // 128 bits

    // ------------------------------------------------ //
    // Step 1. Load Q into Thread Reg.
    constexpr int64_t VEC_LEN = (HEAD_SIZE / VEC_SIZE) / THREAD_GROUP_SIZE;

    static_assert((HEAD_SIZE / THREAD_GROUP_SIZE) % VEC_SIZE == 0);
    static_assert(HEAD_SIZE % THREAD_GROUP_SIZE == 0);
    static_assert(QUANT_GROUP == 8);

    constexpr int64_t QUANT_GROUP_SHIFT = 3;

    // The elements in Q, K, and V will be evenly distributed across each thread group.
    half local_q[VEC_SIZE * VEC_LEN];

    const int64_t warp_id       = threadIdx.x / WARP_SIZE;
    const int64_t warp_lane_id  = threadIdx.x % WARP_SIZE;
    const int64_t group_id      = warp_lane_id / THREAD_GROUP_SIZE;
    const int64_t group_lane_id = warp_lane_id % THREAD_GROUP_SIZE;

    const int64_t cache_offset_s  = PAGE_SIZE <= 0 ? p.cachestarts[batch_idx] : 0;
    const int32_t kv_head_idx     = head_idx / p.num_kv_repeats;

    fp16_t *partial_o       = nullptr;
    fp32_t *partial_log_sum = nullptr;
    int32_t *block_counter  = nullptr;
    if (MULTI_BLOCK > 1) {
        partial_o
            = p.multi_block.partial_out
            + batch_idx * HEAD_SIZE * MULTI_BLOCK
            + head_idx * num_batchs * HEAD_SIZE * MULTI_BLOCK;
        partial_log_sum
            = p.multi_block.partial_log_sum_exp
            + batch_idx * MULTI_BLOCK
            + head_idx * num_batchs * MULTI_BLOCK;
        block_counter
            = p.multi_block.block_counter
            + batch_idx
            + head_idx * num_batchs;
    }

    half *attn_mask = nullptr;
    if (ATTN_MASK) {
        attn_mask = p.attn_mask
                + p.mask_stride_h * head_idx
                + batch_idx * p.mask_stride_s
                + p.kvstarts[batch_idx];
    }

    #pragma unroll
    for (int64_t i = 0; i < VEC_LEN; i++) {
        // copy 128(16 * 8) bits from Q to Local Q

        copy<sizeof(half) * VEC_SIZE>(
            &p.query[
                batch_idx * p.query_stride_s +
                head_idx * HEAD_SIZE +
                (group_lane_id + i * THREAD_GROUP_SIZE) * VEC_SIZE
            ],
            &local_q[i * VEC_SIZE]);
    }

    const int64_t context_len           = p.kvstarts[batch_idx + 1] - p.kvstarts[batch_idx];
    const int64_t context_len_per_block = (context_len + MULTI_BLOCK - 1) / MULTI_BLOCK;
    const int64_t block_context_beg     = block_idx * context_len_per_block;
    const int64_t block_context_len     = context_len >= context_len_per_block * (block_idx + 1) ? context_len_per_block : context_len - block_context_beg;

    __shared__ float tmp_buffer[WPT * HEAD_SIZE];
    float thread_qk_max = -FLT_MAX;
    float partial_exp_sum = 0.0f;

    float local_v[VEC_SIZE * VEC_LEN];
    #pragma unroll
    for(int32_t i = 0; i < VEC_SIZE * VEC_LEN; i++) {
        local_v[i] = 0;
    }

    for (int64_t base_id = warp_id * GPW; base_id < block_context_len; base_id += GPT) {
        float local_v_new[VEC_SIZE * VEC_LEN];
        int8_t local_k_quant[VEC_SIZE * VEC_LEN], local_v_quant[VEC_SIZE * VEC_LEN];
        half local_k_scale[VEC_LEN], local_v_scale[VEC_LEN];
        const int64_t block_context_id = base_id + group_id;

        float qk_dot = 0.0f;

        // all thread groups within a warp must be launched together.
        if (block_context_id < block_context_len) {
            const int64_t cache_token_idx = PAGE_SIZE <= 0
                            ? (cache_offset_s + block_context_beg + block_context_id)
                            : (p.cachestarts[batch_idx * p.cachestarts_stride_b + (block_context_beg + block_context_id) / PAGE_SIZE]
                                + ((block_context_beg + block_context_id) % PAGE_SIZE));
            const int64_t key_offset
                            = cache_token_idx * p.cache_stride_s
                            + p.layer_idx * p.cache_stride_l
                            + p.cache_stride_h * kv_head_idx
                            + group_lane_id * VEC_SIZE;
            const int64_t value_offset = key_offset + p.cache_stride_kv;
            #pragma unroll
            for (int64_t i = 0; i < VEC_LEN; i++) {
                // copy 128(16 * 8) bits from K to Local K
                const int64_t key_idx = key_offset + i * THREAD_GROUP_SIZE * VEC_SIZE;
                copy<sizeof(int8_t) * VEC_SIZE>(&p.cache[key_idx],  &local_k_quant[i * VEC_SIZE]);
                const int64_t key_scale_idx = key_idx >> QUANT_GROUP_SHIFT;
                local_k_scale[i] = p.scale[key_scale_idx];

                // copy 128(16 * 8) bits from V to Local V
                const int64_t value_idx = value_offset + i * THREAD_GROUP_SIZE * VEC_SIZE;
                copy<sizeof(int8_t) * VEC_SIZE>(&p.cache[value_idx],  &local_v_quant[i * VEC_SIZE]);
                const int64_t value_scale_idx = value_idx >> QUANT_GROUP_SHIFT;
                local_v_scale[i] = p.scale[value_scale_idx];

                #pragma unroll
                for(int64_t k = 0; k < VEC_SIZE; k++) {
                    local_k_quant[i * VEC_SIZE + k] += 128;
                    local_v_quant[i * VEC_SIZE + k] += 128;
                }

                half result_k[8];
                uint32_t*      h_k   = reinterpret_cast<uint32_t*>(result_k);
                uint32_t const i8s_k = reinterpret_cast<uint32_t const&>(*(local_k_quant + i * VEC_SIZE));
                asm volatile("prmt.b32 %0,%1,%2,%3;\n" : "=r"(h_k[0]) : "r"(i8s_k), "n"(start_byte_for_fp16), "n"(mask_for_elt_01));
                asm volatile("prmt.b32 %0,%1,%2,%3;\n" : "=r"(h_k[1]) : "r"(i8s_k), "n"(start_byte_for_fp16), "n"(mask_for_elt_23));
                asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h_k[0]) : "r"(h_k[0]), "r"(I8s_TO_F16s_MAGIC_NUM));
                asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h_k[1]) : "r"(h_k[1]), "r"(I8s_TO_F16s_MAGIC_NUM));
                uint32_t*      h_k_2   = reinterpret_cast<uint32_t*>(result_k+4);
                uint32_t const i8s_k_2 = reinterpret_cast<uint32_t const&>(*(local_k_quant + i * VEC_SIZE + 4));
                asm volatile("prmt.b32 %0,%1,%2,%3;\n" : "=r"(h_k_2[0]) : "r"(i8s_k_2), "n"(start_byte_for_fp16), "n"(mask_for_elt_01));
                asm volatile("prmt.b32 %0,%1,%2,%3;\n" : "=r"(h_k_2[1]) : "r"(i8s_k_2), "n"(start_byte_for_fp16), "n"(mask_for_elt_23));
                asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h_k_2[0]) : "r"(h_k_2[0]), "r"(I8s_TO_F16s_MAGIC_NUM));
                asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h_k_2[1]) : "r"(h_k_2[1]), "r"(I8s_TO_F16s_MAGIC_NUM));
                #pragma unroll
                for (int64_t j = 0; j < VEC_SIZE; j++) {
                    qk_dot += __half2float(local_q[i * VEC_SIZE + j]) * __half2float(local_k_scale[i] * result_k[j]);
                }

                half result_v[8];
                uint32_t*      h_v   = reinterpret_cast<uint32_t*>(result_v);
                uint32_t const i8s_v = reinterpret_cast<uint32_t const&>(*(local_v_quant + i * VEC_SIZE));
                asm volatile("prmt.b32 %0,%1,%2,%3;\n" : "=r"(h_v[0]) : "r"(i8s_v), "n"(start_byte_for_fp16), "n"(mask_for_elt_01));
                asm volatile("prmt.b32 %0,%1,%2,%3;\n" : "=r"(h_v[1]) : "r"(i8s_v), "n"(start_byte_for_fp16), "n"(mask_for_elt_23));
                asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h_v[0]) : "r"(h_v[0]), "r"(I8s_TO_F16s_MAGIC_NUM));
                asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h_v[1]) : "r"(h_v[1]), "r"(I8s_TO_F16s_MAGIC_NUM));
                uint32_t*      h_v_2   = reinterpret_cast<uint32_t*>(result_v+4);
                uint32_t const i8s_v_2 = reinterpret_cast<uint32_t const&>(*(local_v_quant + i * VEC_SIZE + 4));
                asm volatile("prmt.b32 %0,%1,%2,%3;\n" : "=r"(h_v_2[0]) : "r"(i8s_v_2), "n"(start_byte_for_fp16), "n"(mask_for_elt_01));
                asm volatile("prmt.b32 %0,%1,%2,%3;\n" : "=r"(h_v_2[1]) : "r"(i8s_v_2), "n"(start_byte_for_fp16), "n"(mask_for_elt_23));
                asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h_v_2[0]) : "r"(h_v_2[0]), "r"(I8s_TO_F16s_MAGIC_NUM));
                asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h_v_2[1]) : "r"(h_v_2[1]), "r"(I8s_TO_F16s_MAGIC_NUM));
                #pragma unroll
                for (int64_t j = 0; j < VEC_SIZE; j++) {
                    local_v_new[i * VEC_SIZE + j] = __half2float(local_v_scale[i] * result_v[j]);
                }
            }
        }

        qk_dot = p.attn_scale * attn_thread_group_reduce_sum<THREAD_GROUP_SIZE>(qk_dot);

        if (block_context_id < block_context_len) {
            if (ATTN_MASK) {
                qk_dot += __half2float(attn_mask[block_context_beg + block_context_id]);
            }
            // Computing inside performs better since using one fma per iteration
            if (qk_dot > thread_qk_max) {
                float logit_scale = exp(thread_qk_max - qk_dot);
                thread_qk_max = qk_dot;
                partial_exp_sum = partial_exp_sum * logit_scale + 1.f;
                #pragma unroll
                for(int32_t i = 0; i < VEC_SIZE * VEC_LEN; i++) {
                    local_v[i] = local_v[i] * logit_scale + local_v_new[i];
                }
            } else {
                float logit_scale = exp(qk_dot - thread_qk_max);
                partial_exp_sum += logit_scale;
                #pragma unroll
                for(int32_t i = 0; i < VEC_SIZE * VEC_LEN; i++) {
                    local_v[i] = local_v[i] + local_v_new[i] * logit_scale;
                }
            }
        }
    }

    // reduce partial_qk_max in thread block and boardcast
    float partial_qk_max = attn_block_reduce_max<WPT, THREAD_GROUP_SIZE>(thread_qk_max, tmp_buffer);

    if (partial_qk_max > thread_qk_max) {
        float logit_scale = exp(thread_qk_max - partial_qk_max);
        partial_exp_sum *= logit_scale;
        #pragma unroll
        for(int32_t i = 0; i < VEC_SIZE * VEC_LEN; i++) {
            local_v[i] *= logit_scale;
        }
    }

    // block reduce sum on partial_exp_sum
    // Warp per thread block must be power-of-2 for reducation, check attn_block_reduce_sum kernel.
    static_assert(WPT == 2 || WPT == 4 || WPT == 8 || WPT == 16 || WPT == 32 || WPT == 64);
    partial_exp_sum = attn_block_reduce_sum<WPT, THREAD_GROUP_SIZE>(partial_exp_sum, &tmp_buffer[WPT]);

    if (MULTI_BLOCK > 1 && threadIdx.x == 0) {
        partial_log_sum[block_idx] = partial_qk_max + log(partial_exp_sum);
    }

    const float inv_sum = __fdividef(1.f, partial_exp_sum + 1e-6f);
    #pragma unroll
    for (int32_t i = 0; i < VEC_SIZE * VEC_LEN; i++) {
        local_v[i] *= inv_sum;
        #pragma unroll
        for (int32_t mask = THREAD_GROUP_SIZE; mask <= WARP_SIZE >> 1; mask = mask << 1) {
            local_v[i] += __shfl_xor_sync(uint32_t(-1), local_v[i], mask);
        }
    }

    // wait for logits to be reused
    __syncthreads();

    constexpr int64_t WORK_WARP = (WPT * THREAD_GROUP_SIZE * VEC_LEN + WARP_SIZE - 1) / WARP_SIZE;
    constexpr int64_t VPT   = 16;
    constexpr int64_t V32PT = 16 / sizeof(float);

    const int32_t v_warp_id  = threadIdx.x % WPT;
    const int32_t v_group_id = (threadIdx.x / WPT) % THREAD_GROUP_SIZE;
    const int32_t v_vec_id   = threadIdx.x / (WPT * THREAD_GROUP_SIZE);

    half local_out[VEC_SIZE];

    // save local_v to shared memory
    if (warp_lane_id < THREAD_GROUP_SIZE) {
        #pragma unroll
        for (int32_t i = 0; i < VEC_LEN * VEC_SIZE; i += V32PT) {
            copy<VPT>(
                &local_v[i],
                &tmp_buffer[
                    i * WPT * THREAD_GROUP_SIZE +
                    warp_lane_id * WPT * V32PT +
                    ((warp_id + warp_lane_id) % WPT) * V32PT]);
        }
    }

    __syncthreads();

    // WPT reduce
    if (warp_id < WORK_WARP) {
        #pragma unroll
        for (int32_t i = 0; i < VEC_SIZE; i+= V32PT) {
            copy<VPT>(
                &tmp_buffer[
                    v_vec_id * VEC_SIZE * WPT * THREAD_GROUP_SIZE +
                    i * WPT * THREAD_GROUP_SIZE +
                    v_group_id * WPT * V32PT +
                    ((v_warp_id + v_group_id) % WPT) * V32PT],
                &local_v[i]);
        }
        #pragma unroll
        for (int32_t i = 0; i < VEC_SIZE; i++) {
            #pragma unroll
            for (int32_t mask = WPT / 2; mask >= 1; mask /= 2) {
                local_v[i] += __shfl_xor_sync(uint32_t(-1), local_v[i], mask);
            }
            local_out[i] = __float2half(local_v[i]);
        }
        if (v_warp_id == 0) {
            half* partial_out = (MULTI_BLOCK == 1)
                    ? &p.output[
                        batch_idx * p.output_stride_s +
                        head_idx * HEAD_SIZE +
                        v_vec_id * THREAD_GROUP_SIZE * VEC_SIZE +
                        v_group_id * VEC_SIZE]
                    : &partial_o[
                        (v_vec_id * THREAD_GROUP_SIZE + v_group_id) * MULTI_BLOCK * VEC_SIZE
                        + block_idx * VEC_SIZE];
            copy<VPT>(local_out, partial_out);
        }
    }

    // Flash decoding
    if (MULTI_BLOCK > 1) {
        __syncthreads();

        bool last_block = false;
        // Make sure every block finishs the partial computation.
        if (threadIdx.x == 0) {
            if (atomicAdd(block_counter, 1) == MULTI_BLOCK - 1) {
                last_block = true;
            }
        }

        // The last block do the final computation.
        if (__syncthreads_or(last_block)) {
            const int64_t multi_block_idx = threadIdx.x % MULTI_BLOCK;

            float local_log_sum_exp = warp_lane_id < MULTI_BLOCK ? partial_log_sum[multi_block_idx] : -FLT_MAX;
            float max_log_sum_exp = local_log_sum_exp;
            # pragma unroll
            for (int32_t mask = MULTI_BLOCK / 2; mask >= 1; mask /= 2) {
                max_log_sum_exp = fmaxf(max_log_sum_exp, __shfl_xor_sync(uint32_t(-1), max_log_sum_exp, mask));
            }
            max_log_sum_exp = __shfl_sync(uint32_t(-1), max_log_sum_exp, 0);

            float local_scale = warp_lane_id < MULTI_BLOCK ? exp(local_log_sum_exp - max_log_sum_exp) : 0.f;
            float scale_sum = local_scale;
            # pragma unroll
            for (int32_t mask = MULTI_BLOCK / 2; mask >= 1; mask /= 2) {
                scale_sum += __shfl_xor_sync(uint32_t(-1), scale_sum, mask);
            }
            scale_sum = __shfl_sync(uint32_t(-1), scale_sum, 0);

            int scale_id = warp_id * MULTI_BLOCK + warp_lane_id;
            if (warp_lane_id < MULTI_BLOCK && scale_id < WARP_SIZE) {
                tmp_buffer[scale_id] = local_scale / scale_sum;
            }
            __syncthreads();

            const int64_t head_dim_idx_base   = threadIdx.x / MULTI_BLOCK * VEC_SIZE;
            const int64_t head_dim_idx_stride = TPB / MULTI_BLOCK * VEC_SIZE;

            #pragma unroll
            for (int64_t head_dim_offset = 0; head_dim_offset < HEAD_SIZE; head_dim_offset += head_dim_idx_stride) {
                int64_t head_dim_idx = head_dim_idx_base + head_dim_offset;
                half final_out[VEC_SIZE];
                local_scale = tmp_buffer[warp_lane_id];
                if (head_dim_idx < HEAD_SIZE) {
                    copy<VEC_SIZE*sizeof(half)>(
                        &partial_o[
                            head_dim_idx * MULTI_BLOCK +
                            multi_block_idx * VEC_SIZE],
                        final_out);
                }

                #pragma unroll
                for (int32_t i = 0; i < VEC_SIZE; i++) {
                    float float_out = __half2float(final_out[i]) * local_scale;
                    # pragma unroll
                    for (int32_t mask = MULTI_BLOCK / 2; mask >= 1; mask /= 2) {
                        float_out += __shfl_xor_sync(uint32_t(-1), float_out, mask);
                    }
                    final_out[i] = __float2half(float_out);
                }

                if (multi_block_idx == 0 && head_dim_idx < HEAD_SIZE) {
                    copy<VPT>(
                        final_out,
                        &p.output[
                            batch_idx * p.output_stride_s +
                            head_idx * HEAD_SIZE +
                            head_dim_idx]);
                }
            }
        }
    }
}

template<
    int32_t HEAD_SIZE,
    int32_t TPB,
    int32_t FULL_GROUP_SIZE,
    int32_t TAIL_GROUP_SIZE,
    bool    IS_TAIL_GROUP
>
__device__ inline
void attn_load_group_query(fp16_t* q_loc, fp16_t* q_glb, fp16_t* q_shm)
{
    constexpr int64_t WARP_SIZE = 32;
    constexpr int64_t MMA_TPG   = 4;
    constexpr int64_t F16PV     = 16 / sizeof(fp16_t);
    constexpr int64_t Q_SIZE    = FULL_GROUP_SIZE * HEAD_SIZE / WARP_SIZE;

    constexpr int64_t VALID_GROUP_SIZE = IS_TAIL_GROUP ? TAIL_GROUP_SIZE : FULL_GROUP_SIZE;

    const int64_t tid           = threadIdx.x;
    const int64_t warp_lane_id  = tid % WARP_SIZE;
    const int64_t group_id      = warp_lane_id / MMA_TPG / 2 + warp_lane_id / MMA_TPG % 2 * 4;
    // const int64_t group_id      = warp_lane_id / MMA_TPG;
    const int64_t group_lane_id = warp_lane_id % MMA_TPG;

    #pragma unroll
    for (int64_t i = 0; i < HEAD_SIZE * VALID_GROUP_SIZE / (TPB * F16PV); i++) {
        int64_t query_group_id  = (tid + i) * F16PV / HEAD_SIZE;
        int64_t head_dim_id     = (tid + i) * F16PV % HEAD_SIZE;
        copy<sizeof(fp16_t) * F16PV>(
            &q_glb[(tid + i) * F16PV],
            &q_shm[query_group_id * (HEAD_SIZE + F16PV) + head_dim_id]);
    }
    if (HEAD_SIZE * VALID_GROUP_SIZE % (TPB * F16PV)) {
        int64_t query_offset
                    = tid * F16PV
                    + HEAD_SIZE * VALID_GROUP_SIZE
                    - HEAD_SIZE * VALID_GROUP_SIZE % (TPB * F16PV);
        if (query_offset < HEAD_SIZE * VALID_GROUP_SIZE) {
            int64_t query_group_id  = query_offset / HEAD_SIZE;
            int64_t head_dim_id     = query_offset % HEAD_SIZE;
            copy<sizeof(fp16_t) * F16PV>(
                &q_glb[query_offset],
                &q_shm[query_group_id * (HEAD_SIZE + F16PV) + head_dim_id]);
        }
    }

    __syncthreads();

    if (IS_TAIL_GROUP) {
        if (group_id < TAIL_GROUP_SIZE) {
            #pragma unroll
            for (int64_t i = 0; i < Q_SIZE; i += F16PV) {
                copy<sizeof(fp16_t) * F16PV>(
                    &q_shm[group_id * (HEAD_SIZE + F16PV) + group_lane_id * Q_SIZE + i],
                    &q_loc[i]);
            }
        } else {
            uint32_t* h_q = reinterpret_cast<uint32_t*>(q_loc);
            #pragma unroll
            for (int64_t i = 0; i < Q_SIZE / 2; i++) {
                h_q[i] = 0;
            }
        }
    } else {
        #pragma unroll
        for (int64_t i = 0; i < Q_SIZE; i += F16PV) {
            copy<sizeof(fp16_t) * F16PV>(
                &q_shm[group_id * (HEAD_SIZE + F16PV) + group_lane_id * Q_SIZE + i],
                &q_loc[i]);
        }
    }
}

template<int32_t LOGITS_SIZE>
__device__ inline
void attn_logits_reorder(fp16_t* dst, fp32_t* src, const int32_t idx1, const int32_t idx2)
{
    constexpr int32_t WARP_SIZE         = 32;
    constexpr int32_t HALF_WARP_SIZE    = WARP_SIZE / 2;
    constexpr int32_t HALF_LOGITS_SIZE  = LOGITS_SIZE / 2;

    const int32_t lane_id = threadIdx.x % WARP_SIZE;

    fp16_t tmp[HALF_LOGITS_SIZE];
    uint32_t* ht = reinterpret_cast<uint32_t*>(tmp);
    uint32_t* hd = reinterpret_cast<uint32_t*>(dst);

    // step 1
    if (lane_id < HALF_WARP_SIZE) {
        #pragma unroll
        for (int32_t i = 0; i < HALF_LOGITS_SIZE; i++) {
            tmp[i] = __float2half(src[2 * i + 1]);
            dst[2 * i] = __float2half(src[2 * i]);
        }
    } else {
        #pragma unroll
        for (int32_t i = 0; i < HALF_LOGITS_SIZE; i++) {
            tmp[i] = __float2half(src[2 * i]);
            dst[2 * i + 1] = __float2half(src[2 * i + 1]);
        }
    }
    #pragma unroll
    for (int32_t i = 0; i < HALF_LOGITS_SIZE / 2; i++) {
        ht[i] = __shfl_sync(uint32_t(-1), ht[i], idx1);
    }

    // step 2
    if (lane_id < HALF_WARP_SIZE) {
        #pragma unroll
        for (int32_t i = 0; i < HALF_LOGITS_SIZE; i++) {
            dst[2 * i + 1] = tmp[i];
        }
    } else {
        #pragma unroll
        for (int32_t i = 0; i < HALF_LOGITS_SIZE; i++) {
            dst[2 * i] = tmp[i];
        }
    }
    #pragma unroll
    for (int32_t i = 0; i < HALF_LOGITS_SIZE; i++) {
        hd[i] = __shfl_sync(uint32_t(-1), hd[i], idx2);
    }
}

template<int32_t WPT, int32_t GROUP_SIZE, int32_t VEC_SIZE>
__device__ inline
void attn_block_reduce_group_max(fp32_t *dst, fp32_t *src, fp32_t *shared_mem)
{
    constexpr int32_t WARP_SIZE = 32;
    constexpr int32_t LOAD_SIZE = WPT * VEC_SIZE * GROUP_SIZE / WARP_SIZE;
    const int32_t lane_id = threadIdx.x % WARP_SIZE;
    const int32_t warp_id = threadIdx.x / WARP_SIZE;

    if (lane_id < GROUP_SIZE) {
        copy<sizeof(fp32_t) * VEC_SIZE>(src, &shared_mem[warp_id * GROUP_SIZE * VEC_SIZE + lane_id * VEC_SIZE]);
    }
    __syncthreads();

    fp32_t tmp[LOAD_SIZE];
    #pragma unroll
    for (int32_t i = 0; i < LOAD_SIZE; i += VEC_SIZE) {
        copy<sizeof(fp32_t) * VEC_SIZE>(&shared_mem[i * WARP_SIZE + lane_id * VEC_SIZE], &tmp[i]);
    }

    #pragma unroll
    for (int32_t i = 1; i < LOAD_SIZE / VEC_SIZE; i++) {
        #pragma unroll
        for (int32_t j = 0; j < VEC_SIZE; j++) {
            tmp[j] = fmaxf(tmp[j], tmp[i * VEC_SIZE + j]);
        }
    }

    #pragma unroll
    for (int32_t i = 0; i < VEC_SIZE; i++) {
        dst[i] = tmp[i];
        #pragma unroll
        for (int32_t mask = GROUP_SIZE; mask < WARP_SIZE; mask <<= 1) {
            dst[i] = fmaxf(dst[i], __shfl_xor_sync(uint32_t(-1), dst[i], mask));
        }
    }
}

template<int32_t WPT, int32_t GROUP_SIZE, int32_t VEC_SIZE>
__device__ inline
void attn_block_reduce_group_sum(fp32_t *reducing, fp32_t *shared_mem)
{
    constexpr int32_t WARP_SIZE = 32;
    constexpr int32_t LOAD_SIZE = WPT * VEC_SIZE * GROUP_SIZE / WARP_SIZE;
    const int32_t lane_id = threadIdx.x % WARP_SIZE;
    const int32_t warp_id = threadIdx.x / WARP_SIZE;

    #pragma unroll
    for (int32_t i = 0; i < VEC_SIZE; i++) {
        #pragma unroll
        for (int32_t mask = GROUP_SIZE; mask < WARP_SIZE; mask <<= 1) {
            reducing[i] += __shfl_xor_sync(uint32_t(-1), reducing[i], mask);
        }
    }

    if (lane_id < GROUP_SIZE) {
        copy<sizeof(fp32_t) * VEC_SIZE>(reducing, &shared_mem[warp_id * GROUP_SIZE * VEC_SIZE + lane_id * VEC_SIZE]);
    }
    __syncthreads();

    fp32_t tmp[LOAD_SIZE];
    #pragma unroll
    for (int32_t i = 0; i < LOAD_SIZE; i += VEC_SIZE) {
        copy<sizeof(fp32_t) * VEC_SIZE>(&shared_mem[i * WARP_SIZE + lane_id * VEC_SIZE], &tmp[i]);
    }

    #pragma unroll
    for (int32_t i = 1; i < LOAD_SIZE / VEC_SIZE; i++) {
        #pragma unroll
        for (int32_t j = 0; j < VEC_SIZE; j++) {
            tmp[j] += tmp[i * VEC_SIZE + j];
        }
    }
    #pragma unroll
    for (int32_t i = 0; i < VEC_SIZE; i++) {
        reducing[i] = tmp[i];
        #pragma unroll
        for (int32_t mask = GROUP_SIZE; mask < WARP_SIZE; mask <<= 1) {
            reducing[i] += __shfl_xor_sync(uint32_t(-1), reducing[i], mask);
        }
    }
}


template<
    int32_t HEAD_SIZE,
    int32_t TPB,
    int32_t QUANT_GROUP,
    int32_t QUERY_GROUP,
    int32_t MULTI_BLOCK,    // do flash decoding if more than 1
    bool    ATTN_MASK,
    int32_t PAGE_SIZE>
__global__
void dynamic_batching_decoding_group_query_cache_attention_fp16_kernel(dynamic_batching_decoding_cache_attention_kernel_param p)
{
    static constexpr uint32_t I8s_TO_F16s_MAGIC_NUM = 0x64806480;
    static constexpr uint32_t mask_for_elt_01       = 0x5150;
    static constexpr uint32_t mask_for_elt_23       = 0x5352;
    static constexpr uint32_t start_byte_for_fp16   = 0x64646464;

    constexpr int64_t WARP_SIZE = 32;                   // warp size
    constexpr int64_t WPT       = TPB / WARP_SIZE;      // warp per thread block

    constexpr int64_t VEC_WIDTH = 16;                           // 128 bits
    constexpr int64_t I8PV      = VEC_WIDTH / sizeof(int8_t);   // num per vector for int_8
    constexpr int64_t F16PV     = VEC_WIDTH / sizeof(fp16_t);   // num per vector for fp16_t
    constexpr int64_t F32PV     = VEC_WIDTH / sizeof(fp32_t);   // num per vector for fp32_t

    // HMMA.16816.F32
    constexpr int64_t MMA_M     = 16;
    constexpr int64_t MMA_N     = 8;
    constexpr int64_t MMA_K     = 16;

    constexpr int64_t MMA_TPG   = 4;                    // every 4 threads handle one major line(K) in MMA
    constexpr int64_t MMA_GPT   = 2;                    // each thread handles 2 query group
    constexpr int64_t MMA_GPW   = WARP_SIZE / MMA_TPG;  // thread group per warp

    constexpr int64_t CONTEXT_STRIDE    = MMA_M;
    constexpr int64_t FULL_GROUP_SIZE   = MMA_N;
    constexpr int64_t TAIL_GROUP_SIZE   = QUERY_GROUP % FULL_GROUP_SIZE;
    constexpr int64_t GROUP_BLOCK_SIZE  = (QUERY_GROUP + FULL_GROUP_SIZE - 1) / FULL_GROUP_SIZE;

    constexpr int64_t MMA_QPT   = MMA_N * MMA_K / WARP_SIZE;
    constexpr int64_t MMA_KPT   = MMA_M * MMA_K / WARP_SIZE;
    constexpr int64_t MMA_LPT   = MMA_M * MMA_N / WARP_SIZE;
    constexpr int64_t MMA_VPT   = MMA_M * MMA_K / WARP_SIZE;
    constexpr int64_t MMA_OPT   = MMA_M * MMA_N / WARP_SIZE;
    constexpr int64_t MMA_LPG   = MMA_LPT / MMA_GPT;
    constexpr int64_t MMA_VPG   = MMA_VPT / MMA_GPT;
    constexpr int64_t MMA_OPG   = MMA_OPT / MMA_GPT;

    constexpr int64_t Q_SIZE    = MMA_N * HEAD_SIZE / WARP_SIZE;
    constexpr int64_t K_SIZE    = MMA_M * HEAD_SIZE / WARP_SIZE;
    constexpr int64_t L_SIZE    = MMA_M * MMA_N / WARP_SIZE;
    constexpr int64_t V_SIZE    = MMA_M * HEAD_SIZE / WARP_SIZE;
    constexpr int64_t O_SIZE    = MMA_N * HEAD_SIZE / WARP_SIZE;

    constexpr int64_t K_PER_CONTEXT     = HEAD_SIZE / MMA_TPG;
    constexpr int64_t V_PER_CONTEXT     = HEAD_SIZE / MMA_GPW;
    constexpr int64_t O_PER_QUERY       = HEAD_SIZE / MMA_GPW;
    constexpr int64_t KS_PER_CONTEXT    = K_PER_CONTEXT / QUANT_GROUP;
    constexpr int64_t VS_PER_CONTEXT    = (V_PER_CONTEXT + QUANT_GROUP - 1) / QUANT_GROUP;

    constexpr int64_t K_LOAD_STRIDE
                        = (K_PER_CONTEXT % I8PV == 0) ? I8PV
                        : (K_PER_CONTEXT % (I8PV / 2) == 0) ? (I8PV / 2)
                        : (I8PV / 4);
    constexpr int64_t V_LOAD_STRIDE
                        = (V_PER_CONTEXT % I8PV == 0) ? I8PV
                        : (V_PER_CONTEXT % (I8PV / 2) == 0) ? (I8PV / 2)
                        : (I8PV / 4);
    constexpr int64_t KS_LOAD_STRIDE
                        = (KS_PER_CONTEXT % F16PV == 0) ? F16PV
                        : (KS_PER_CONTEXT % (F16PV / 2) == 0) ? (F16PV / 2)
                        : (KS_PER_CONTEXT % (F16PV / 4) == 0) ? (F16PV / 4)
                        : 1;
    constexpr int64_t VS_LOAD_STRIDE
                        = (V_PER_CONTEXT % QUANT_GROUP) ? 1
                        : (VS_PER_CONTEXT % F16PV == 0) ? F16PV
                        : (VS_PER_CONTEXT % (F16PV / 2) == 0) ? (F16PV / 2)
                        : (VS_PER_CONTEXT % (F16PV / 4) == 0) ? (F16PV / 4)
                        : 1;

    constexpr int64_t VALID_LOGIT_SIZE  = (QUERY_GROUP <= FULL_GROUP_SIZE / MMA_GPT) ? 1 : 2;
    constexpr int64_t VALID_REDUCE_HEAD = (QUERY_GROUP <= FULL_GROUP_SIZE) ? QUERY_GROUP : FULL_GROUP_SIZE;
    constexpr int64_t HEAD_PER_REDUCE
                        = (WPT * HEAD_SIZE > 4096) ? 1
                        : (WPT * HEAD_SIZE > 2048) ? 2
                        : (WPT * HEAD_SIZE > 1024) ? 4
                        : 8;
    constexpr int64_t THREAD_PER_REDUCE = WPT * HEAD_PER_REDUCE * HEAD_SIZE / F16PV;
    constexpr int64_t THREAD_PER_HEAD   = HEAD_SIZE / F16PV;

    const int64_t num_batchs    = gridDim.y;
    const int64_t batch_idx     = blockIdx.y;
    const int64_t block_idx     = blockIdx.z;
    const int32_t qo_head_base  = (QUERY_GROUP <= FULL_GROUP_SIZE)
                    ? blockIdx.x * QUERY_GROUP
                    : blockIdx.x / GROUP_BLOCK_SIZE * QUERY_GROUP + blockIdx.x % GROUP_BLOCK_SIZE * FULL_GROUP_SIZE;
    const int32_t kv_head_idx   = (QUERY_GROUP <= FULL_GROUP_SIZE)
                    ? blockIdx.x
                    : blockIdx.x / GROUP_BLOCK_SIZE;

    const int64_t tid           = threadIdx.x;
    const int64_t warp_id       = tid / WARP_SIZE;
    const int64_t warp_lane_id  = tid % WARP_SIZE;
    const int64_t group_id      = warp_lane_id / MMA_TPG;
    const int64_t group_lane_id = warp_lane_id % MMA_TPG;

    const bool is_tail_group = (QUERY_GROUP < FULL_GROUP_SIZE)
                    ? true
                    : ((TAIL_GROUP_SIZE > 0) && (blockIdx.x % GROUP_BLOCK_SIZE == GROUP_BLOCK_SIZE - 1));

    const int64_t logit_reorder_id1 = (warp_lane_id + WARP_SIZE / 2) % WARP_SIZE;
    const int64_t logit_reorder_id2
                    = warp_lane_id / MMA_TPG % 2 * 16
                    + warp_lane_id / MMA_TPG / 2
                    + warp_lane_id % MMA_TPG * 4;

    const int64_t context_len           = p.kvstarts[batch_idx + 1] - p.kvstarts[batch_idx];
    const int64_t context_len_per_block = (context_len + MULTI_BLOCK - 1) / MULTI_BLOCK;
    const int64_t block_context_beg     = block_idx * context_len_per_block;
    const int64_t block_context_len     = (context_len >= context_len_per_block * (block_idx + 1))
                    ? context_len_per_block
                    : context_len - block_context_beg;

    const int64_t cache_token_base = (PAGE_SIZE <= 0)
                    ? p.cachestarts[batch_idx] + block_context_beg
                    : 0;
    const int64_t cahce_offset
                    = cache_token_base * p.cache_stride_s
                    + p.layer_idx * p.cache_stride_l
                    + kv_head_idx * p.cache_stride_h;

    fp16_t *q_glb       = p.query + batch_idx * p.query_stride_s + qo_head_base * HEAD_SIZE;
    int8_t *k_glb       = p.cache + cahce_offset + group_lane_id * K_PER_CONTEXT;
    int8_t *v_glb       = p.cache + cahce_offset + group_id * V_PER_CONTEXT + p.cache_stride_kv;
    fp16_t *k_scale_glb = p.scale + (cahce_offset + group_lane_id * K_PER_CONTEXT) / QUANT_GROUP;
    fp16_t *v_scale_glb = p.scale + (cahce_offset + group_id * V_PER_CONTEXT + p.cache_stride_kv) / QUANT_GROUP;
    fp16_t *o_glb       = p.output + batch_idx * p.output_stride_s + qo_head_base * HEAD_SIZE;

    fp16_t *partial_o       = nullptr;
    fp32_t *partial_log_sum = nullptr;
    int32_t *block_counter  = nullptr;
    if (MULTI_BLOCK > 1) {
        partial_o
            = p.multi_block.partial_out
            + batch_idx * HEAD_SIZE * MULTI_BLOCK
            + qo_head_base * num_batchs * HEAD_SIZE * MULTI_BLOCK;
        partial_log_sum
            = p.multi_block.partial_log_sum_exp
            + batch_idx * MULTI_BLOCK
            + qo_head_base * num_batchs * MULTI_BLOCK;
        block_counter
            = p.multi_block.block_counter
            + batch_idx
            + qo_head_base * num_batchs;
    }

    fp16_t *attn_mask = nullptr;
    if (ATTN_MASK) {
        attn_mask
            = p.attn_mask
            + qo_head_base * p.mask_stride_h
            + batch_idx * p.mask_stride_s
            + p.kvstarts[batch_idx]
            + block_context_beg;
    }

    __shared__ fp32_t tmp_buffer[HEAD_PER_REDUCE * (WPT * HEAD_SIZE + F32PV)];
    fp16_t *q_shm = reinterpret_cast<fp16_t*>(tmp_buffer);
    fp16_t q_loc[Q_SIZE];
    fp32_t warp_qk_max[MMA_LPG], warp_exp_sum[MMA_LPG], warp_o[O_SIZE];

    if (is_tail_group) {
        attn_load_group_query<HEAD_SIZE, TPB, FULL_GROUP_SIZE, TAIL_GROUP_SIZE, true>(q_loc, q_glb, q_shm);
    } else {
        attn_load_group_query<HEAD_SIZE, TPB, FULL_GROUP_SIZE, TAIL_GROUP_SIZE, false>(q_loc, q_glb, q_shm);
    }

    #pragma unroll
    for (int64_t i = 0; i < MMA_LPG; i++) {
        warp_qk_max[i]  = -FLT_MAX;
        warp_exp_sum[i] = 0.f;
    }
    #pragma unroll
    for (int64_t i = 0; i < O_SIZE; i++) {
        warp_o[i] = 0.f;
    }

    for (int64_t base_id = warp_id * CONTEXT_STRIDE; base_id < block_context_len; base_id += WPT * CONTEXT_STRIDE) {
        fp32_t qk_dot[L_SIZE], tile_qk_max[MMA_LPG];
        fp16_t k_loc_reordered[K_SIZE], v_loc_reordered[V_SIZE];

        #pragma unroll
        for (int64_t i = 0; i < L_SIZE; i++) {
            qk_dot[i] = 0.f;
        }

        #pragma unroll
        for (int64_t k_context_group = 0; k_context_group < MMA_GPT; k_context_group++) {
            const int64_t k_context_id = base_id + group_id + k_context_group * MMA_GPW;
            if (k_context_id < block_context_len) {
                int8_t k_quant[K_PER_CONTEXT];
                fp16_t k_scale_loc[KS_PER_CONTEXT], k_loc[K_PER_CONTEXT];
                const int64_t cache_token_idx = (PAGE_SIZE <= 0)
                                ? k_context_id
                                : (p.cachestarts[batch_idx * p.cachestarts_stride_b + (block_context_beg + k_context_id) / PAGE_SIZE]
                                    + ((block_context_beg + k_context_id) % PAGE_SIZE));
                const int64_t key_offset = cache_token_idx * p.cache_stride_s;

                #pragma unroll
                for (int64_t i = 0; i < KS_PER_CONTEXT; i += KS_LOAD_STRIDE) {
                    copy<sizeof(fp16_t) * KS_LOAD_STRIDE>(&k_scale_glb[key_offset / QUANT_GROUP + i],  &k_scale_loc[i]);
                }

                #pragma unroll
                for (int64_t i = 0; i < K_PER_CONTEXT; i += K_LOAD_STRIDE) {
                    const int64_t key_idx = key_offset + i;
                    copy<sizeof(int8_t) * K_LOAD_STRIDE>(&k_glb[key_idx],  &k_quant[i]);

                    #pragma unroll
                    for (int64_t j = 0; j < K_LOAD_STRIDE; j++) {
                        k_quant[i + j] += 128;
                    }

                    #pragma unroll
                    for (int64_t j = 0; j < K_LOAD_STRIDE; j += 4) {
                        uint32_t*      h_k   = reinterpret_cast<uint32_t*>(k_loc + i + j);
                        uint32_t const i8s_k = reinterpret_cast<uint32_t const&>(*(k_quant + i + j));
                        asm volatile("prmt.b32 %0,%1,%2,%3;\n" : "=r"(h_k[0]) : "r"(i8s_k), "n"(start_byte_for_fp16), "n"(mask_for_elt_01));
                        asm volatile("prmt.b32 %0,%1,%2,%3;\n" : "=r"(h_k[1]) : "r"(i8s_k), "n"(start_byte_for_fp16), "n"(mask_for_elt_23));
                        asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h_k[0]) : "r"(h_k[0]), "r"(I8s_TO_F16s_MAGIC_NUM));
                        asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h_k[1]) : "r"(h_k[1]), "r"(I8s_TO_F16s_MAGIC_NUM));
                    }

                    #pragma unroll
                    for (int64_t j = 0; j < K_LOAD_STRIDE; j += 2) {
                        k_loc_reordered[(k_context_group + i + j) * 2]     = k_loc[i + j]     * k_scale_loc[(i + j) / QUANT_GROUP];
                        k_loc_reordered[(k_context_group + i + j) * 2 + 1] = k_loc[i + j + 1] * k_scale_loc[(i + j + 1) / QUANT_GROUP];
                    }
                }
            } else {
                uint32_t* h_k = reinterpret_cast<uint32_t*>(k_loc_reordered);
                #pragma unroll
                for (int64_t i = 0; i < K_PER_CONTEXT / 2; i++) {
                    h_k[k_context_group + i * 2] = 0;
                }
            }
        }

        #pragma unroll
        for (int64_t i = 0; i < HEAD_SIZE / MMA_K; i++) {
            uint32_t* A = reinterpret_cast<uint32_t*>(k_loc_reordered + i * MMA_KPT);
            uint32_t* B = reinterpret_cast<uint32_t*>(q_loc + i * MMA_QPT);
            uint32_t* D = reinterpret_cast<uint32_t*>(qk_dot);
            asm volatile(
                "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 {%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%0,%1,%2,%3};\n"
                    : "+r"(D[0]), "+r"(D[1]), "+r"(D[2]), "+r"(D[3])
                    : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]), "r"(B[0]), "r"(B[1])
            );
        }

        #pragma unroll
        for (int64_t i = 0; i < VALID_LOGIT_SIZE; i++) {
            const int64_t query_group_idx = group_lane_id + i * MMA_TPG;
            tile_qk_max[i] = -FLT_MAX;
            #pragma unroll
            for (int64_t j = 0; j < MMA_GPT; j++) {
                const int64_t l_context_id = base_id + group_id + j * MMA_GPW;
                if (l_context_id < block_context_len) {
                    qk_dot[j * MMA_LPG + i] *= p.attn_scale;
                    if (ATTN_MASK) {
                        int64_t valid_group_idx = (is_tail_group && query_group_idx >= TAIL_GROUP_SIZE) ? 0 : query_group_idx;
                        qk_dot[j * MMA_LPG + i] += __half2float(attn_mask[valid_group_idx * p.mask_stride_h + l_context_id]);
                    }
                    tile_qk_max[i] = fmaxf(tile_qk_max[i], qk_dot[j * MMA_LPG + i]);
                }
            }
            #pragma unroll
            for (int32_t mask = MMA_TPG; mask < WARP_SIZE; mask <<= 1) {
                tile_qk_max[i] = fmaxf(tile_qk_max[i], __shfl_xor_sync(uint32_t(-1), tile_qk_max[i], mask));
            }
            if (tile_qk_max[i] > warp_qk_max[i]) {
                fp32_t logit_scale = exp(warp_qk_max[i] - tile_qk_max[i]);
                #pragma unroll
                for (int64_t j = 0; j < O_SIZE; j += MMA_OPT) {
                    #pragma unroll
                    for (int64_t k = 0; k < MMA_GPT; k++) {
                        warp_o[j + k * MMA_OPG + i] *= logit_scale;
                    }
                }
                warp_exp_sum[i] *= logit_scale;
                warp_qk_max[i] = tile_qk_max[i];
            }
            #pragma unroll
            for (int64_t j = 0; j < MMA_GPT; j++) {
                const int64_t l_context_id = base_id + group_id + j * MMA_GPW;
                if (l_context_id < block_context_len) {
                    qk_dot[j * MMA_LPG + i] = exp(qk_dot[j * MMA_LPG + i] - warp_qk_max[i]);
                    warp_exp_sum[i] += qk_dot[j * MMA_LPG + i];
                }
            }
        }

        fp16_t logit[L_SIZE];
        attn_logits_reorder<L_SIZE>(logit, qk_dot, logit_reorder_id1, logit_reorder_id2);

        #pragma unroll
        for (int64_t v_context_group = 0; v_context_group < MMA_VPG; v_context_group++) {
            int64_t v_context_id = base_id + v_context_group * MMA_TPG + group_lane_id;
            if (v_context_id < block_context_len) {
                int8_t v_quant[V_PER_CONTEXT];
                fp16_t v_scale_loc[VS_PER_CONTEXT], v_scale_loc2[VS_PER_CONTEXT * 2], v_loc[V_PER_CONTEXT];
                const int64_t cache_token_idx = (PAGE_SIZE <= 0)
                                ? v_context_id
                                : (p.cachestarts[batch_idx * p.cachestarts_stride_b + (block_context_beg + v_context_id) / PAGE_SIZE]
                                    + ((block_context_beg + v_context_id) % PAGE_SIZE));
                const int64_t value_offset = cache_token_idx * p.cache_stride_s;

                #pragma unroll
                for (int64_t i = 0; i < VS_PER_CONTEXT; i += VS_LOAD_STRIDE) {
                    copy<sizeof(fp16_t) * VS_LOAD_STRIDE>(&v_scale_glb[value_offset / QUANT_GROUP + i],  &v_scale_loc[i]);
                }

                if (V_PER_CONTEXT % QUANT_GROUP) {
                    if (group_id % 2) {
                        #pragma unroll
                        for (int64_t i = 1; i < VS_PER_CONTEXT; i++) {
                            v_scale_loc2[2 * i] = v_scale_loc[i];
                            v_scale_loc2[2 * i - 1] = v_scale_loc[i];
                        }
                        v_scale_loc2[0] = v_scale_loc[0];
                    } else {
                        #pragma unroll
                        for (int64_t i = 0; i < VS_PER_CONTEXT - 1; i++) {
                            v_scale_loc2[2 * i] = v_scale_loc[i];
                            v_scale_loc2[2 * i + 1] = v_scale_loc[i];
                        }
                        v_scale_loc2[2 * VS_PER_CONTEXT - 2] = v_scale_loc[VS_PER_CONTEXT - 1];
                    }
                }

                #pragma unroll
                for (int64_t i = 0; i < V_PER_CONTEXT; i += V_LOAD_STRIDE) {
                    const int64_t value_idx = value_offset + i;
                    copy<sizeof(int8_t) * V_LOAD_STRIDE>(&v_glb[value_idx],  &v_quant[i]);

                    #pragma unroll
                    for (int64_t j = 0; j < V_LOAD_STRIDE; j++) {
                        v_quant[i + j] += 128;
                    }

                    #pragma unroll
                    for (int64_t j = 0; j < V_LOAD_STRIDE; j += 4) {
                        uint32_t*      h_v   = reinterpret_cast<uint32_t*>(v_loc + i + j);
                        uint32_t const i8s_v = reinterpret_cast<uint32_t const&>(*(v_quant + i + j));
                        asm volatile("prmt.b32 %0,%1,%2,%3;\n" : "=r"(h_v[0]) : "r"(i8s_v), "n"(start_byte_for_fp16), "n"(mask_for_elt_01));
                        asm volatile("prmt.b32 %0,%1,%2,%3;\n" : "=r"(h_v[1]) : "r"(i8s_v), "n"(start_byte_for_fp16), "n"(mask_for_elt_23));
                        asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h_v[0]) : "r"(h_v[0]), "r"(I8s_TO_F16s_MAGIC_NUM));
                        asm volatile("sub.f16x2 %0, %1, %2;\n" : "=r"(h_v[1]) : "r"(h_v[1]), "r"(I8s_TO_F16s_MAGIC_NUM));
                    }

                    #pragma unroll
                    for (int64_t j = 0; j < V_LOAD_STRIDE; j += 2) {
                        fp16_t v_scale = (V_PER_CONTEXT % QUANT_GROUP)
                                    ? v_scale_loc2[(i + j) / (QUANT_GROUP / 2)]
                                    : v_scale_loc[(i + j) / QUANT_GROUP];
                        v_loc_reordered[2 * v_context_group - v_context_group % 2 + (i + j) * MMA_VPG]     = v_loc[i + j]     * v_scale;
                        v_loc_reordered[2 * v_context_group - v_context_group % 2 + (i + j) * MMA_VPG + 2] = v_loc[i + j + 1] * v_scale;
                    }
                }
            } else {
                uint16_t* h_v = reinterpret_cast<uint16_t*>(v_loc_reordered);
                #pragma unroll
                for (int64_t i = 0; i < V_PER_CONTEXT; i += MMA_GPT) {
                    h_v[2 * v_context_group - v_context_group % 2 + i * MMA_VPG]     = 0;
                    h_v[2 * v_context_group - v_context_group % 2 + i * MMA_VPG + 2] = 0;
                }
            }
        }

        #pragma unroll
        for (int64_t i = 0; i < HEAD_SIZE / MMA_M; i++) {
            uint32_t* A = reinterpret_cast<uint32_t*>(v_loc_reordered + i * MMA_VPT);
            uint32_t* B = reinterpret_cast<uint32_t*>(logit);
            uint32_t* D = reinterpret_cast<uint32_t*>(warp_o + i * MMA_OPT);
            asm volatile(
                "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 {%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%0,%1,%2,%3};\n"
                    : "+r"(D[0]), "+r"(D[1]), "+r"(D[2]), "+r"(D[3])
                    : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]), "r"(B[0]), "r"(B[1])
            );
        }
    }

    fp32_t *reduce_buffer = reinterpret_cast<fp32_t*>(q_shm + (HEAD_SIZE + F16PV) * FULL_GROUP_SIZE);
    fp32_t block_qk_max[MMA_LPG], warp_o_reordered[O_SIZE];
    attn_block_reduce_group_max<WPT, MMA_TPG, VALID_LOGIT_SIZE>(block_qk_max, warp_qk_max, reduce_buffer);

    #pragma unroll
    for (int64_t i = 0; i < VALID_LOGIT_SIZE; i++) {
        if (block_qk_max[i] > warp_qk_max[i]) {
            fp32_t logit_scale = exp(warp_qk_max[i] - block_qk_max[i]);
            warp_exp_sum[i] *= logit_scale;
            #pragma unroll
            for (int64_t j = 0; j < O_SIZE; j += MMA_LPG) {
                warp_o[i + j] *= logit_scale;
            }
        }
    }

    attn_block_reduce_group_sum<WPT, MMA_TPG, VALID_LOGIT_SIZE>(warp_exp_sum, &reduce_buffer[WPT * FULL_GROUP_SIZE]);

    if (MULTI_BLOCK > 1 && tid < MMA_TPG) {
        if (is_tail_group) {
            #pragma unroll
            for (int64_t i = 0; i < MMA_LPG; i++) {
                int64_t head_offset_loc = group_lane_id + i * MMA_TPG;
                if (head_offset_loc < TAIL_GROUP_SIZE) {
                    partial_log_sum[head_offset_loc * num_batchs * MULTI_BLOCK + block_idx]
                        = block_qk_max[i] + log(warp_exp_sum[i]);
                }
            }
        } else {
            #pragma unroll
            for (int64_t i = 0; i < MMA_LPG; i++) {
                partial_log_sum[(group_lane_id + i * MMA_TPG) * num_batchs * MULTI_BLOCK + block_idx]
                    = block_qk_max[i] + log(warp_exp_sum[i]);
            }
        }
    }

    #pragma unroll
    for (int64_t i = 0; i < VALID_LOGIT_SIZE; i++) {
        const fp32_t inv_sum = __fdividef(1.f, warp_exp_sum[i] + 1e-6f);
        #pragma unroll
        for (int64_t j = 0; j < O_SIZE / MMA_LPG; j++) {
            warp_o_reordered[i * O_SIZE / MMA_LPG + j] = warp_o[i + j * MMA_LPG] * inv_sum;
        }
    }

    const int64_t reduce_warp_id  = tid % WPT;

    #pragma unroll
    for (int64_t head_reduce = 0; head_reduce < VALID_REDUCE_HEAD; head_reduce += HEAD_PER_REDUCE) {
        // wait for logits to be reused
        __syncthreads();

        if (is_tail_group) {
            #pragma unroll
            for (int64_t i = 0; i < VALID_LOGIT_SIZE; i++) {
                int64_t head_offset_glb = group_lane_id + i * MMA_TPG;
                if (head_offset_glb >= head_reduce && head_offset_glb < head_reduce + HEAD_PER_REDUCE && head_offset_glb < TAIL_GROUP_SIZE) {
                    int64_t head_offset_shm = head_offset_glb - head_reduce;

                    #pragma unroll
                    for (int64_t j = 0; j < O_PER_QUERY; j += F32PV) {
                        copy<sizeof(fp32_t) * F32PV>(
                            &warp_o_reordered[i * O_PER_QUERY + j],
                            &tmp_buffer[
                                head_offset_shm * (WPT * HEAD_SIZE + F32PV)
                                + group_id * O_PER_QUERY * WPT
                                + j * WPT
                                + warp_id * F32PV]);
                    }
                }
            }
        } else {
            #pragma unroll
            for (int64_t i = 0; i < HEAD_PER_REDUCE / FULL_GROUP_SIZE + 1; i++) {
                if (
                    (HEAD_PER_REDUCE == 8)
                    || (HEAD_PER_REDUCE == 4)
                    || (HEAD_PER_REDUCE == 2 && (group_lane_id == head_reduce % 4 || group_lane_id == head_reduce % 4 + 1))
                    || (HEAD_PER_REDUCE == 1 && group_lane_id == head_reduce % 4)
                ) {
                    int64_t head_offset_shm
                                = (HEAD_PER_REDUCE == 8) ? (group_lane_id + i * MMA_TPG)
                                : (HEAD_PER_REDUCE == 4) ? group_lane_id
                                : (HEAD_PER_REDUCE == 2) ? (group_lane_id % HEAD_PER_REDUCE)
                                : 0;
                    int64_t head_offset_loc = (HEAD_PER_REDUCE == 8) ? i : (head_reduce / 4);

                    #pragma unroll
                    for (int64_t j = 0; j < O_PER_QUERY; j += F32PV) {
                        copy<sizeof(fp32_t) * F32PV>(
                            &warp_o_reordered[head_offset_loc * O_PER_QUERY + j],
                            &tmp_buffer[
                                head_offset_shm * (WPT * HEAD_SIZE + F32PV)
                                + group_id * O_PER_QUERY * WPT
                                + j * WPT
                                + warp_id * F32PV]);
                    }
                }
            }
        }

        __syncthreads();
        #pragma unroll
        for (int64_t i = 0; i < THREAD_PER_REDUCE; i += TPB) {
            fp16_t block_o_loc[F16PV];

            const int64_t head_dim_idx      = (i + tid) / WPT % THREAD_PER_HEAD * F16PV;
            const int64_t head_offset_shm   = (i + tid) / WPT / THREAD_PER_HEAD;
            const int64_t head_offset_glb   = head_offset_shm + head_reduce;

            bool is_reduce_thread = is_tail_group
                            ? (head_offset_glb < TAIL_GROUP_SIZE)
                            : (THREAD_PER_REDUCE % TPB == 0 || head_offset_shm < HEAD_PER_REDUCE);

            if (is_reduce_thread) {
                #pragma unroll
                for (int64_t j = 0; j < F16PV; j += F32PV) {
                    copy<sizeof(fp32_t) * F32PV>(
                        &tmp_buffer[
                            head_offset_shm * (WPT * HEAD_SIZE + F32PV)
                            + head_dim_idx * WPT
                            + j * WPT
                            + reduce_warp_id * F32PV],
                        &warp_o[j]);
                }
            }

            #pragma unroll
            for (int64_t j = 0; j < F16PV; j++) {
                #pragma unroll
                for (int32_t mask = WPT / 2; mask > 0; mask >>= 1) {
                    warp_o[j] += __shfl_xor_sync(uint32_t(-1), warp_o[j], mask);
                }
                block_o_loc[j] = __float2half(warp_o[j]);
            }

            if (reduce_warp_id == 0 && is_reduce_thread) {
                fp16_t *block_o_glb = (MULTI_BLOCK == 1)
                            ? &o_glb[head_offset_glb * HEAD_SIZE + head_dim_idx]
                            : &partial_o[
                                head_offset_glb * num_batchs * HEAD_SIZE * MULTI_BLOCK
                                + head_dim_idx * MULTI_BLOCK
                                + block_idx * F16PV];
                copy<sizeof(fp16_t) * F16PV>(block_o_loc, block_o_glb);
            }
        }
    }

    // Flash decoding
    if (MULTI_BLOCK > 1) {
        __syncthreads();

        bool last_block = false;
        // Make sure every block finishs the partial computation.
        if (tid == 0) {
            if (atomicAdd(block_counter, 1) == MULTI_BLOCK - 1) {
                last_block = true;
            }
        }

        // The last block do the final computation.
        if (__syncthreads_or(last_block)) {
            const int64_t multi_block_idx   = tid % MULTI_BLOCK;
            const int64_t head_dim_base     = tid / MULTI_BLOCK * F16PV;
            const int64_t head_dim_stride   = TPB / MULTI_BLOCK * F16PV;

            const int64_t block_reduce_head = (QUERY_GROUP > FULL_GROUP_SIZE) && (TAIL_GROUP_SIZE > 0) && (blockIdx.x % GROUP_BLOCK_SIZE == GROUP_BLOCK_SIZE - 1)
                                ? TAIL_GROUP_SIZE
                                : VALID_REDUCE_HEAD;

            for (int64_t head_offset = 0; head_offset < block_reduce_head; head_offset++) {
                // get max block log sum exp
                fp32_t local_log_sum_exp = (warp_lane_id < MULTI_BLOCK)
                            ? partial_log_sum[head_offset * num_batchs * MULTI_BLOCK + multi_block_idx]
                            : -FLT_MAX;
                fp32_t max_log_sum_exp = local_log_sum_exp;
                # pragma unroll
                for (int32_t mask = MULTI_BLOCK / 2; mask >= 1; mask /= 2) {
                    max_log_sum_exp = fmaxf(max_log_sum_exp, __shfl_xor_sync(uint32_t(-1), max_log_sum_exp, mask));
                }
                max_log_sum_exp = __shfl_sync(uint32_t(-1), max_log_sum_exp, 0);

                // update scale
                fp32_t local_scale = (warp_lane_id < MULTI_BLOCK)
                            ? exp(local_log_sum_exp - max_log_sum_exp)
                            : 0.f;
                fp32_t scale_sum = local_scale;
                # pragma unroll
                for (int32_t mask = MULTI_BLOCK / 2; mask >= 1; mask /= 2) {
                    scale_sum += __shfl_xor_sync(uint32_t(-1), scale_sum, mask);
                }
                scale_sum = __shfl_sync(uint32_t(-1), scale_sum, 0);

                int scale_id = warp_id * MULTI_BLOCK + warp_lane_id;
                if (warp_lane_id < MULTI_BLOCK && scale_id < WARP_SIZE) {
                    tmp_buffer[scale_id] = local_scale / scale_sum;
                }
                __syncthreads();

                // block ouput reduce
                #pragma unroll
                for (int64_t head_dim_offset = 0; head_dim_offset < HEAD_SIZE; head_dim_offset += head_dim_stride) {
                    int64_t head_dim_idx = head_dim_base + head_dim_offset;
                    fp16_t final_out[F16PV];
                    local_scale = tmp_buffer[warp_lane_id];
                    if (head_dim_idx < HEAD_SIZE) {
                        copy<sizeof(fp16_t) * F16PV>(
                            &partial_o[
                                head_offset * num_batchs * HEAD_SIZE * MULTI_BLOCK +
                                head_dim_idx * MULTI_BLOCK +
                                multi_block_idx * F16PV],
                            final_out);
                    }

                    #pragma unroll
                    for (int64_t i = 0; i < F16PV; i++) {
                        fp32_t float_out = __half2float(final_out[i]) * local_scale;
                        # pragma unroll
                        for (int32_t mask = MULTI_BLOCK / 2; mask >= 1; mask /= 2) {
                            float_out += __shfl_xor_sync(uint32_t(-1), float_out, mask);
                        }
                        final_out[i] = __float2half(float_out);
                    }

                    if (multi_block_idx == 0 && head_dim_idx < HEAD_SIZE) {
                        copy<sizeof(fp16_t) * F16PV>(
                            final_out,
                            &o_glb[head_offset * HEAD_SIZE + head_dim_idx]);
                    }
                }
            }
        }
    }
}

template<
    int32_t HEAD_SIZE,
    int32_t TPB,
    bool    ATTN_MASK,
    bool    DO_MULTI_BLOCK,
    int32_t PAGE_SIZE>
ppl::common::RetCode dynamic_batching_decoding_cache_attention(
    const hipStream_t stream,
    const dynamic_batching_multi_head_cache_attention::config &cfg,
    const dynamic_batching_decoding_cache_attention_kernel_param &p
)
{
    const int64_t RAW_SHM_SIZE = 48 * 1024;

    const int32_t QUANT_GROUP = 8;
    const int32_t MULTI_BLOCK = DO_MULTI_BLOCK ? 32 / (HEAD_SIZE / 64) : 1;
    const int32_t THREAD_GROUP_SIZE = HEAD_SIZE / 64 * 4;
    const int32_t FULL_QUERY_GROUP  = 8;

    int64_t decoding_shm_size = 0;
    int64_t query_group = cfg.num_heads;
    auto kernel_fn = dynamic_batching_decoding_cache_infinity_attention_fp16_kernel<HEAD_SIZE, THREAD_GROUP_SIZE, TPB, QUANT_GROUP, MULTI_BLOCK, ATTN_MASK, PAGE_SIZE>;
    if (cfg.decoding_algo == decoding_algo::INFINITY_GQCA) {
        query_group = (cfg.num_kv_repeats + FULL_QUERY_GROUP - 1) / FULL_QUERY_GROUP * (cfg.num_heads / cfg.num_kv_repeats);
        switch (cfg.num_kv_repeats) {
            case 4:
                kernel_fn = dynamic_batching_decoding_group_query_cache_attention_fp16_kernel<HEAD_SIZE, TPB, QUANT_GROUP, 4, MULTI_BLOCK, ATTN_MASK, PAGE_SIZE>;
                break;
            case 6:
                kernel_fn = dynamic_batching_decoding_group_query_cache_attention_fp16_kernel<HEAD_SIZE, TPB, QUANT_GROUP, 6, MULTI_BLOCK, ATTN_MASK, PAGE_SIZE>;
                break;
            case 8:
                kernel_fn = dynamic_batching_decoding_group_query_cache_attention_fp16_kernel<HEAD_SIZE, TPB, QUANT_GROUP, 8, MULTI_BLOCK, ATTN_MASK, PAGE_SIZE>;
                break;
            case 16:
                kernel_fn = dynamic_batching_decoding_group_query_cache_attention_fp16_kernel<HEAD_SIZE, TPB, QUANT_GROUP, 16, MULTI_BLOCK, ATTN_MASK, PAGE_SIZE>;
                break;
            default:
                LOG(ERROR) << "group query cache flash decoding attention do not support query group size " << cfg.num_kv_repeats;
                return ppl::common::RC_UNSUPPORTED;
        }
    } else if (cfg.decoding_algo == decoding_algo::INFINITY_MHCA) {
        kernel_fn = dynamic_batching_decoding_cache_infinity_attention_fp16_kernel<HEAD_SIZE, THREAD_GROUP_SIZE, TPB, QUANT_GROUP, MULTI_BLOCK, ATTN_MASK, PAGE_SIZE>;
    } else if (cfg.decoding_algo == decoding_algo::SHAREMEM_MHCA) {
        kernel_fn = dynamic_batching_decoding_cache_sharemem_attention_fp16_kernel<HEAD_SIZE, THREAD_GROUP_SIZE, TPB, QUANT_GROUP, MULTI_BLOCK, ATTN_MASK, PAGE_SIZE>;

        const int32_t WARP_SIZE = 32;
        const int32_t WPT = TPB / WARP_SIZE;
        const int32_t reduce_shm_size = TPB / WARP_SIZE * sizeof(float);
        const int64_t max_multi_block_kvlen = (cfg.max_kvlen * sizeof(float) + cfg.decoding_multi_block_size - 1) / cfg.decoding_multi_block_size;
        decoding_shm_size = max(max_multi_block_kvlen, WPT * HEAD_SIZE * sizeof(float));

        if (decoding_shm_size > RAW_SHM_SIZE - reduce_shm_size) {
            auto cuda_err = hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_fn), hipFuncAttributeMaxDynamicSharedMemorySize, decoding_shm_size);
            if (cuda_err == hipErrorInvalidValue) {
                LOG(ERROR) << "this gpu does not have enough shared-memory cache flash decoding attention requires";
                return ppl::common::RC_UNSUPPORTED;
            }
        }
    } else {
        LOG(ERROR) << "unknown decoding cache attention algo: " << cfg.decoding_algo;
        return ppl::common::RC_INVALID_VALUE;
    }

    const dim3 grid_size = {
        (unsigned int)query_group,
        (unsigned int)cfg.decoding_batches,
        (unsigned int)cfg.decoding_multi_block_size};
    kernel_fn<<<grid_size, TPB, decoding_shm_size, stream>>>(p);

    return ppl::common::RC_SUCCESS;
}

template<int32_t TPB, bool DO_MULTI_BLOCK, int32_t PAGE_SIZE>
ppl::common::RetCode dynamic_batching_decoding_cache_attention(
    const hipStream_t stream,
    const dynamic_batching_multi_head_cache_attention::config &cfg,
    const dynamic_batching_decoding_cache_attention_kernel_param &p
)
{
    ppl::common::RetCode status = ppl::common::RC_UNSUPPORTED;
    if (p.attn_mask) {
        switch (cfg.head_dim) {
            case 64:
                status = dynamic_batching_decoding_cache_attention<64, TPB, true, DO_MULTI_BLOCK, PAGE_SIZE>(stream, cfg, p);
                break;
            case 96:
                status = dynamic_batching_decoding_cache_attention<96, TPB, true, DO_MULTI_BLOCK, PAGE_SIZE>(stream, cfg, p);
                break;
            case 128:
                status = dynamic_batching_decoding_cache_attention<128, TPB, true, DO_MULTI_BLOCK, PAGE_SIZE>(stream, cfg, p);
                break;
            case 256:
                status = dynamic_batching_decoding_cache_attention<256, TPB, true, DO_MULTI_BLOCK, PAGE_SIZE>(stream, cfg, p);
                break;
            default:
                LOG(ERROR) << "cache flash decoding attention do not support head dim " << cfg.head_dim;
        }
    } else {
        switch (cfg.head_dim) {
            case 64:
                status = dynamic_batching_decoding_cache_attention<64, TPB, false, DO_MULTI_BLOCK, PAGE_SIZE>(stream, cfg, p);
                break;
            case 96:
                status = dynamic_batching_decoding_cache_attention<96, TPB, false, DO_MULTI_BLOCK, PAGE_SIZE>(stream, cfg, p);
                break;
            case 128:
                status = dynamic_batching_decoding_cache_attention<128, TPB, false, DO_MULTI_BLOCK, PAGE_SIZE>(stream, cfg, p);
                break;
            case 256:
                status = dynamic_batching_decoding_cache_attention<256, TPB, false, DO_MULTI_BLOCK, PAGE_SIZE>(stream, cfg, p);
                break;
            default:
                LOG(ERROR) << "cache flash decoding attention do not support head dim " << cfg.head_dim;
        }
    }

    return status;
}

ppl::common::RetCode dynamic_batching_multi_head_cache_attention::forward_kvstore(
    const hipStream_t stream)
{
    struct dynamic_batching_kv_cache_quantize_kernel_param kv_store_p{0};
    kv_store_p.current_key = (half*)cfg.current_key;
    kv_store_p.current_value =  (half*)cfg.current_value;
    kv_store_p.seqstarts = (int64_t*)cfg.seqstarts;
    kv_store_p.cachestarts = (int64_t*)cfg.cachestarts;
    kv_store_p.start_pos = (int64_t*)cfg.start_pos;
    kv_store_p.num_layer = cfg.num_layer;
    kv_store_p.layer_idx = cfg.layer_idx;
    kv_store_p.num_kv_heads = cfg.num_kv_heads;
    kv_store_p.head_dim = cfg.head_dim;
    kv_store_p.page_size = cfg.page_size;
    kv_store_p.current_key_stride_s = cfg.k_stride_s;
    kv_store_p.current_value_stride_s = cfg.v_stride_s;
    kv_store_p.cache_stride_s = cfg.cache_stride_s;
    kv_store_p.cache_stride_l = cfg.cache_stride_l;
    kv_store_p.cache_stride_h = cfg.cache_stride_h;
    kv_store_p.cache_stride_kv = cfg.cache_stride_kv;
    kv_store_p.cachestarts_stride_b = cfg.cachestarts_stride_b;
    kv_store_p.cache = (int8_t*)cfg.cache;
    kv_store_p.scale = (half*)cfg.scale;
    {
        constexpr int64_t TPB = 256;
        constexpr int64_t VPT = 8;

        if (cfg.head_dim % VPT != 0) {
            LOG(ERROR) << "head_dim must be aligned with " << VPT << ", currently get " << cfg.head_dim;
            return ppl::common::RC_UNSUPPORTED;
        }

        dim3 grid(cfg.max_seqlen, cfg.batch, (cfg.num_kv_heads * cfg.head_dim / VPT + TPB - 1) / TPB);
        if (cfg.cache_mode == 1) {
            dynamic_batching_kv_cache_quantize_kernel<VPT, TPB, UNIFORM_PAGE_SIZE><<<grid, TPB, 0, stream>>>(kv_store_p);
        } else if (cfg.cache_mode == 0) {
            dynamic_batching_kv_cache_quantize_kernel<VPT, TPB, 0><<<grid, TPB, 0, stream>>>(kv_store_p);
        } else {
            LOG(ERROR) << "invalid cache_mode " << cfg.cache_mode;
            return ppl::common::RC_UNSUPPORTED;
        }
    }

    return ppl::common::RC_SUCCESS;
}

ppl::common::RetCode dynamic_batching_multi_head_cache_attention::forward_decode(
    const hipStream_t stream)
{
    struct dynamic_batching_decoding_cache_attention_kernel_param deocde_p{0};
    deocde_p.query = (half*)cfg.query;
    deocde_p.attn_mask = (half*)cfg.attn_mask;
    deocde_p.output = (half*)cfg.output;
    deocde_p.cache = (int8_t*)cfg.cache;
    deocde_p.scale = (half*)cfg.scale;
    deocde_p.cachestarts = (int64_t*)cfg.cachestarts;
    deocde_p.kvstarts = (int64_t*)cfg.kvstarts;
    deocde_p.attn_scale = cfg.attn_scale;
    deocde_p.layer_idx = cfg.layer_idx;
    deocde_p.num_kv_repeats = cfg.num_kv_repeats;
    deocde_p.page_size = cfg.page_size;
    deocde_p.query_stride_s = cfg.q_stride_s;
    deocde_p.output_stride_s = cfg.o_stride_s;
    deocde_p.mask_stride_s = cfg.mask_stride_s;
    deocde_p.mask_stride_h = cfg.mask_stride_h;
    deocde_p.cache_stride_s = cfg.cache_stride_s;
    deocde_p.cache_stride_l = cfg.cache_stride_l;
    deocde_p.cache_stride_h = cfg.cache_stride_h;
    deocde_p.cache_stride_kv = cfg.cache_stride_kv;
    deocde_p.cachestarts_stride_b = cfg.cachestarts_stride_b;

    if(cfg.decoding_batches > 0) {
        ppl::common::RetCode status = ppl::common::RC_UNSUPPORTED;
        if (cfg.decoding_multi_block_size > 1) {
            deocde_p.multi_block.partial_out           = (half*)cfg.workspace;
            deocde_p.multi_block.partial_log_sum_exp   = reinterpret_cast<float*>((char*)cfg.workspace
                + cfg.decoding_multi_block_partial_out_size);
            deocde_p.multi_block.block_counter         = reinterpret_cast<int32_t*>((char*)cfg.workspace
                + cfg.decoding_multi_block_partial_out_size
                + cfg.decoding_multi_block_partial_log_sum_exp_size);
            hipMemsetAsync(deocde_p.multi_block.block_counter, 0, cfg.decoding_multi_block_counter_size, stream);
            if (cfg.cache_mode == 1) {
                status = dynamic_batching_decoding_cache_attention<256, true, UNIFORM_PAGE_SIZE>(stream, cfg, deocde_p);
            } else {
                status = dynamic_batching_decoding_cache_attention<256, true, 0>(stream, cfg, deocde_p);
            }
        } else if (cfg.decoding_threads_per_block == 256) {
            if (cfg.cache_mode == 1) {
                status = dynamic_batching_decoding_cache_attention<256, false, UNIFORM_PAGE_SIZE>(stream, cfg, deocde_p);
            } else {
                status = dynamic_batching_decoding_cache_attention<256, false, 0>(stream, cfg, deocde_p);
            }
        } else if (cfg.decoding_threads_per_block == 512) {
            if (cfg.cache_mode == 1) {
                status = dynamic_batching_decoding_cache_attention<512, false, UNIFORM_PAGE_SIZE>(stream, cfg, deocde_p);
            } else {
                status = dynamic_batching_decoding_cache_attention<512, false, 0>(stream, cfg, deocde_p);
            }
        } else if (cfg.decoding_threads_per_block == 1024) {
            if (cfg.cache_mode == 1) {
                status = dynamic_batching_decoding_cache_attention<1024, false, UNIFORM_PAGE_SIZE>(stream, cfg, deocde_p);
            } else {
                status = dynamic_batching_decoding_cache_attention<1024, false, 0>(stream, cfg, deocde_p);
            }
        }
        if (status != ppl::common::RC_SUCCESS) {
            LOG(ERROR) << "unsupported decoding_multi_block_size and decoding_threads_per_block";
            return ppl::common::RC_UNSUPPORTED;
        }
    }

    return ppl::common::RC_SUCCESS;
}

ppl::common::RetCode dynamic_batching_multi_head_cache_attention::forward_prefill(
    const hipStream_t stream)
{
    if (cfg.prefill_batches > 0) {
        const void* prefill_seqstart_q = ((int64_t*)cfg.seqstarts) + cfg.decoding_batches;
        const void* prefill_seqstart_k = ((int64_t*)cfg.kvstarts) + cfg.decoding_batches;

        if (cfg.enable_cache_prefill) {
            int64_t quant_bit = 8, quant_group = 8;

            const void* prefill_key   = ((int8_t*)cfg.cache) + cfg.layer_idx * cfg.cache_stride_l;
            const void* prefill_value = ((int8_t*)prefill_key) + cfg.cache_stride_kv;

            const void* prefill_key_scale   = ((fp16_t*)cfg.scale) + cfg.layer_idx * (int64_t)(cfg.cache_stride_l / quant_group);
            const void* prefill_value_scale = ((fp16_t*)prefill_key_scale) + (int64_t)(cfg.cache_stride_kv / quant_group);

            return llm::cuda::flash_attn2::flash_attn2_paged_fmha(
                stream,
                *cfg.device_prop,
                cfg.datatype,
                cfg.query,
                prefill_key,
                prefill_value,
                cfg.attn_mask,
                prefill_seqstart_q,
                prefill_seqstart_k,
                cfg.cachestarts,
                nullptr,
                prefill_key_scale,
                prefill_value_scale,
                cfg.alibi_slopes,
                cfg.prefill_batches,
                0, cfg.q_stride_s, cfg.head_dim,
                cfg.cache_stride_s, cfg.cache_stride_s, cfg.cache_stride_h,
                cfg.cache_stride_s, cfg.cache_stride_s, cfg.cache_stride_h,
                0, cfg.mask_stride_s, cfg.mask_stride_h,
                0,
                cfg.o_stride_s,
                cfg.max_seqlen,
                cfg.max_kvlen,
                cfg.num_heads,
                cfg.num_kv_heads,
                cfg.head_dim,
                (int64_t)cfg.is_causal,
                cfg.page_size,
                cfg.cachestarts_stride_b,
                quant_bit,
                quant_group,
                cfg.attn_scale,
                cfg.output);
        } else {
            return llm::cuda::flash_attn2::flash_attn2_fmha(
                stream,
                *cfg.device_prop,
                cfg.datatype,
                cfg.query,
                cfg.current_key,
                cfg.current_value,
                cfg.attn_mask,
                prefill_seqstart_q,
                prefill_seqstart_k,
                nullptr,
                nullptr,
                cfg.alibi_slopes,
                cfg.prefill_batches,
                0, cfg.q_stride_s, cfg.head_dim,
                0, cfg.k_stride_s, cfg.head_dim,
                0, cfg.v_stride_s, cfg.head_dim,
                0, cfg.mask_stride_s, cfg.mask_stride_h,
                0,
                cfg.o_stride_s,
                cfg.max_seqlen,
                cfg.max_kvlen,
                cfg.num_heads,
                cfg.num_kv_heads,
                cfg.head_dim,
                (int64_t)cfg.is_causal,
                0,
                0,
                cfg.attn_scale,
                cfg.output);
        }
    } else {
        return ppl::common::RC_SUCCESS;
    }
}

ppl::common::RetCode dynamic_batching_multi_head_cache_attention::forward(
    const hipStream_t stream)
{
    ppl::common::RetCode ret;

    ret = forward_kvstore(stream);
    if (ret != ppl::common::RC_SUCCESS) {
        LOG(ERROR) << "kv_stroe stage failed.";
    }

    ret = forward_decode(stream);
    if (ret != ppl::common::RC_SUCCESS) {
        LOG(ERROR) << "decode stage failed.";
    }

    ret = forward_prefill(stream);
    if (ret != ppl::common::RC_SUCCESS) {
        LOG(ERROR) << "prefill stage failed.";
    }

    return ret;
}

}}}}}
