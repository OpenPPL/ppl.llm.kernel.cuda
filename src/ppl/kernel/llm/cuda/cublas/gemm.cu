#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include "ppl/kernel/llm/cuda/cublas/gemm.h"
#include "ppl/common/log.h"

#include <hip/hip_fp16.h>

namespace ppl { namespace kernel { namespace llm { namespace cuda { namespace cublas {

#if (CUDART_VERSION < 11000)
template <typename T>
__global__ void cublas_gemm_add_bias_kernel(int64_t num_elems, int64_t bias_elems, T* output, const T* bias) {
    const int64_t output_idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (output_idx >= num_elems)
        return;

    output[output_idx] += bias[output_idx % bias_elems];
}
#endif

#define CUBLAS_CHECK_RC(X) do { \
        hipblasStatus_t __status = (X); \
        if (__status != HIPBLAS_STATUS_SUCCESS) { \
            LOG(ERROR) << "cublasLt failed: " << cublasLtGetStatusString(__status); \
            return ppl::common::RC_DEVICE_RUNTIME_ERROR; \
        } \
    } while (0)

ppl::common::RetCode gemm(
    const hipStream_t stream,
    const hipblasLtHandle_t& cublaslt_handle,
    const hipblasLtMatmulAlgo_t* algo,
    const bool transa,
    const int64_t lda,
    const ppl::common::datatype_t typea,
    const void* A,
    const bool transb,
    const int64_t ldb,
    const ppl::common::datatype_t typeb,
    const void* B,
    const void* bias,
    const int64_t M,
    const int64_t N,
    const int64_t K,
    const float alpha,
    const float beta,
    const int64_t workspace_size,
    void* workspace,
    const int64_t ldc,
    const ppl::common::datatype_t typec,
    void* C)
{
    if (typea != ppl::common::DATATYPE_FLOAT16) {
        LOG(ERROR) << "only support fp16 A matrix";
        return ppl::common::RC_UNSUPPORTED;
    }
    if (typeb != ppl::common::DATATYPE_FLOAT16) {
        LOG(ERROR) << "only support fp16 B matrix";
        return ppl::common::RC_UNSUPPORTED;
    }
    if (typec != ppl::common::DATATYPE_FLOAT16) {
        LOG(ERROR) << "only support fp16 C matrix";
        return ppl::common::RC_UNSUPPORTED;
    }

    hipblasLtMatmulDesc_t operationDesc = nullptr;
    hipblasLtMatrixLayout_t Adesc = nullptr, Bdesc = nullptr, Cdesc = nullptr;

    hipblasOperation_t cublas_transa = transa == true ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t cublas_transb = transb == true ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipDataType scaleType = HIP_R_16F;

#if (CUDART_VERSION >= 11000)
    hipblasComputeType_t computeType = HIPBLAS_COMPUTE_32F;
    CUBLAS_CHECK_RC(hipblasLtMatmulDescCreate(&operationDesc, computeType, scaleType));
#else
    hipDataType computeType = scaleType;
    CUBLAS_CHECK_RC(hipblasLtMatmulDescCreate(&operationDesc, computeType));
#endif

    // exchange A & B to col-major
    CUBLAS_CHECK_RC(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &cublas_transb, sizeof(cublas_transb)));
    CUBLAS_CHECK_RC(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &cublas_transa, sizeof(cublas_transa)));

#if (CUDART_VERSION >= 11000)
    if (bias != nullptr) {
        hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_BIAS;
        CUBLAS_CHECK_RC(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue)));
        CUBLAS_CHECK_RC(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(void*)));
    }
#endif
    // create matrix descriptors, we are good with the details here so no need to set any extra attributes
    CUBLAS_CHECK_RC(hipblasLtMatrixLayoutCreate(&Adesc, scaleType, cublas_transa == HIPBLAS_OP_N ? K : M, cublas_transa == HIPBLAS_OP_N ? M : K, lda));
    CUBLAS_CHECK_RC(hipblasLtMatrixLayoutCreate(&Bdesc, scaleType, cublas_transb == HIPBLAS_OP_N ? N : K, cublas_transb == HIPBLAS_OP_N ? K : N, ldb));
    CUBLAS_CHECK_RC(hipblasLtMatrixLayoutCreate(&Cdesc, scaleType, N, M, ldc));

    CUBLAS_CHECK_RC(hipblasLtMatmul(
        cublaslt_handle,
        operationDesc,
        (const void*)(&alpha),
        B,
        Bdesc,
        A,
        Adesc,
        (const void*)(&beta),
        C,
        Cdesc,
        C,
        Cdesc,
        algo,
        workspace,
        workspace_size,
        stream));

#if (CUDART_VERSION < 11000)
    if (bias != nullptr) {
        const int64_t num_elems = M * N;
        const int64_t block_size = 128;
        const int64_t blocks = (num_elems + block_size - 1) / block_size;
        dim3 grid_size(blocks, 1, 1);
        cublas_gemm_add_bias_kernel<half><<<grid_size, block_size, 0, stream>>>(M * N, N, (half*)C, (const half*)bias);
    }
#endif

    // descriptors are no longer needed as all GPU work was already enqueued
    if (Cdesc) CUBLAS_CHECK_RC(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) CUBLAS_CHECK_RC(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) CUBLAS_CHECK_RC(hipblasLtMatrixLayoutDestroy(Adesc));
    if (operationDesc) CUBLAS_CHECK_RC(hipblasLtMatmulDescDestroy(operationDesc));
    
    return ppl::common::RC_SUCCESS;
}

ppl::common::RetCode int8_gemm(
    const hipStream_t stream,
    const hipblasLtHandle_t& cublaslt_handle,
    const hipblasLtMatmulAlgo_t* algo,
    const bool transa, // must be false
    const int64_t lda, // transa ? M : K;
    const ppl::common::datatype_t typea, // int8
    const void* A, // int8
    const bool transb, // must be true
    const int64_t ldb, // transb ? K : N;
    const ppl::common::datatype_t typeb, // int8
    const void* B, // int8
    const void* bias, // int32
    const int64_t M,
    const int64_t N,
    const int64_t K,
    const int32_t alpha, // int32-C need
    const int32_t beta, // int32-C need
    const int64_t workspace_size,
    void* workspace,
    const int64_t ldc, // N
    const ppl::common::datatype_t typec, // int32
    void* C) // int32
{
    if (typea != ppl::common::DATATYPE_INT8) {
        LOG(ERROR) << "only support int8 A matrix";
        return ppl::common::RC_UNSUPPORTED;
    }
    if (typeb != ppl::common::DATATYPE_INT8) {
        LOG(ERROR) << "only support int8 B matrix";
        return ppl::common::RC_UNSUPPORTED;
    }
    if (typec != ppl::common::DATATYPE_INT32) {
        LOG(ERROR) << "only support int32 C matrix";
        return ppl::common::RC_UNSUPPORTED;
    }

    hipblasLtMatmulDesc_t operationDesc = nullptr;
    hipblasLtMatrixLayout_t Adesc = nullptr, Bdesc = nullptr, Cdesc = nullptr;

    hipblasOperation_t cublas_transa = transa == true ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t cublas_transb = transb == true ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipDataType scaleType = HIP_R_32I;
    hipDataType abType = HIP_R_8I;
    hipDataType cType = HIP_R_32I;

#if (CUDART_VERSION >= 11000)
    hipblasComputeType_t computeType = HIPBLAS_COMPUTE_32I;
    CUBLAS_CHECK_RC(hipblasLtMatmulDescCreate(&operationDesc, computeType, scaleType));
#else
    hipDataType computeType = scaleType;
    CUBLAS_CHECK_RC(hipblasLtMatmulDescCreate(&operationDesc, computeType));
#endif

    // exchange A & B to col-major
    CUBLAS_CHECK_RC(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &cublas_transb, sizeof(cublas_transb)));
    CUBLAS_CHECK_RC(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &cublas_transa, sizeof(cublas_transa)));

#if (CUDART_VERSION >= 11000)
    if (bias != nullptr) {
        hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_BIAS;
        CUBLAS_CHECK_RC(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue)));
        CUBLAS_CHECK_RC(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(void*)));
    }
#endif
    // create matrix descriptors, we are good with the details here so no need to set any extra attributes
    CUBLAS_CHECK_RC(hipblasLtMatrixLayoutCreate(&Adesc, abType, cublas_transa == HIPBLAS_OP_N ? K : M, cublas_transa == HIPBLAS_OP_N ? M : K, lda));
    CUBLAS_CHECK_RC(hipblasLtMatrixLayoutCreate(&Bdesc, abType, cublas_transb == HIPBLAS_OP_N ? N : K, cublas_transb == HIPBLAS_OP_N ? K : N, ldb));
    CUBLAS_CHECK_RC(hipblasLtMatrixLayoutCreate(&Cdesc, cType, N, M, ldc));

    CUBLAS_CHECK_RC(hipblasLtMatmul(
        cublaslt_handle,
        operationDesc,
        (const void*)(&alpha),
        B,
        Bdesc,
        A,
        Adesc,
        (const void*)(&beta),
        C,
        Cdesc,
        C,
        Cdesc,
        nullptr,
        workspace,
        workspace_size,
        stream));

#if (CUDART_VERSION < 11000)
    if (bias != nullptr) {
        const int64_t num_elems = M * N;
        const int64_t block_size = 128;
        const int64_t blocks = (num_elems + block_size - 1) / block_size;
        dim3 grid_size(blocks, 1, 1);
        cublas_gemm_add_bias_kernel<int32_t><<<grid_size, block_size, 0, stream>>>(M * N, N, (int32_t*)C, (const int32_t*)bias);
    }
#endif

    // descriptors are no longer needed as all GPU work was already enqueued
    if (Cdesc) CUBLAS_CHECK_RC(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) CUBLAS_CHECK_RC(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) CUBLAS_CHECK_RC(hipblasLtMatrixLayoutDestroy(Adesc));
    if (operationDesc) CUBLAS_CHECK_RC(hipblasLtMatmulDescDestroy(operationDesc));
    
    return ppl::common::RC_SUCCESS;
}

}}}}}
