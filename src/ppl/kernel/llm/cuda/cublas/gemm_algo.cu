#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include "ppl/kernel/llm/cuda/cublas/gemm_algo.h"

#include "ppl/common/log.h"

#include <algorithm>
#include <map>

namespace ppl { namespace kernel { namespace llm { namespace cuda { namespace cublas {

#define CUBLAS_CHECK_RC(X) do { \
        hipblasStatus_t __status = (X); \
        if (__status != HIPBLAS_STATUS_SUCCESS) { \
            LOG(ERROR) << "cublasLt failed: " << cublasLtGetStatusString(__status); \
            return {ppl::common::RC_DEVICE_RUNTIME_ERROR, hipblasLtMatmulAlgo_t{}}; \
        } \
    } while (0)

std::pair<ppl::common::RetCode, hipblasLtMatmulAlgo_t> find_best_algo(
    const hipStream_t     stream,
    const hipblasLtHandle_t&lightHandle,
    const std::vector<int>&banned_algo_ids,
    hipblasLtMatmulDesc_t   computeDesc,
    const void*            alpha,
    const void*            A,
    hipblasLtMatrixLayout_t Adesc,
    const void*            B,
    hipblasLtMatrixLayout_t Bdesc,
    const void*            beta,
    const void*            C,
    hipblasLtMatrixLayout_t Cdesc,
    void*                  D,
    hipblasLtMatrixLayout_t Ddesc,
    const int64_t          workspace_size,
    void*                  workspace)
{
    size_t returnSize;
    int32_t pointer_mode;
    hipblasLtMatmulDescGetAttribute(
        computeDesc, HIPBLASLT_MATMUL_DESC_POINTER_MODE, &pointer_mode, sizeof(pointer_mode), &returnSize);

    std::vector<hipblasLtMatmulHeuristicResult_t> heuristics(200);
    hipblasLtMatmulPreference_t preference;
    CUBLAS_CHECK_RC(hipblasLtMatmulPreferenceCreate(&preference));
    CUBLAS_CHECK_RC(cublasLtMatmulPreferenceInit(preference));
    CUBLAS_CHECK_RC(hipblasLtMatmulPreferenceSetAttribute(
        preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspace_size, sizeof(workspace_size)));
#if (CUBLAS_VERSION) <= 12000
    uint32_t pointer_mode_mask = 0;
    CUBLAS_CHECK_RC(hipblasLtMatmulPreferenceSetAttribute(
        preference, CUBLASLT_MATMUL_PREF_EPILOGUE_MASK, &pointer_mode_mask, sizeof(pointer_mode_mask)));
#endif

    int return_count = 0;
    auto ret = hipblasLtMatmulAlgoGetHeuristic(lightHandle,
                                              computeDesc,
                                              Adesc,
                                              Bdesc,
                                              Cdesc,
                                              Ddesc,
                                              preference,
                                              heuristics.size(),
                                              heuristics.data(),
                                              &return_count);
    heuristics.resize(return_count);

    std::map<int, std::vector<float>> algo_results;
    for (const auto& heuristic : heuristics) {
        hipblasLtMatmulAlgo_t algo = heuristic.algo;
        int32_t algo_id;
        cublasLtMatmulAlgoConfigGetAttribute(&algo, CUBLASLT_ALGO_CONFIG_ID, &algo_id, sizeof(algo_id), &returnSize);

        if (std::find(banned_algo_ids.begin(), banned_algo_ids.end(), algo_id) != banned_algo_ids.end())
            continue;

        hipEvent_t start_event, stop_event;
        hipEventCreate(&start_event);
        hipEventCreate(&stop_event);

        for (int i = 0; i < 11; i++) {
            float duration_ms;
            hipEventRecord(start_event, stream);
            CUBLAS_CHECK_RC(hipblasLtMatmul(lightHandle,
                                            computeDesc,
                                            alpha,
                                            A,
                                            Adesc,
                                            B,
                                            Bdesc,
                                            beta,
                                            C,
                                            Cdesc,
                                            D,
                                            Ddesc,
                                            &algo,
                                            workspace,
                                            workspace_size,
                                            stream));
            hipEventRecord(stop_event, stream);
            hipEventSynchronize(stop_event);
            hipEventElapsedTime(&duration_ms, start_event, stop_event);

            algo_results[algo_id].push_back(duration_ms);
        }

        hipEventDestroy(start_event);
        hipEventDestroy(stop_event);

        std::sort(algo_results[algo_id].begin(), algo_results[algo_id].end());
    }

    hipblasLtMatmulHeuristicResult_t result;
    float best_time = INFINITY;
    for (const auto& heuristic : heuristics) {
        hipblasLtMatmulAlgo_t algo = heuristic.algo;
        int32_t algo_id;
        cublasLtMatmulAlgoConfigGetAttribute(&algo, CUBLASLT_ALGO_CONFIG_ID, &algo_id, sizeof(algo_id), &returnSize);
        const auto& results = algo_results[algo_id];

        if (results.size() > 0 && results[5] < best_time) {
            best_time = results[5];
            result = heuristic;
        }
    }

    CUBLAS_CHECK_RC(hipblasLtMatmulPreferenceDestroy(preference));

    return {best_time != INFINITY ? ppl::common::RC_SUCCESS : ppl::common::RC_NOT_FOUND, result.algo};
}

}}}}}
